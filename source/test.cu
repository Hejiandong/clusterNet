#include <stdio.h>
#include <hipblas.h>
#include <util.cuh>
#include <basicOps.cuh>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <util.cuh>
#include <clusterNet.cuh>


void run_neural_network()
{
  Matrix X = read_csv("/home/tim/Downloads/mnist_full_X.csv");
  Matrix y = read_csv("/home/tim/Downloads/mnist_full_y.csv");

  //w1 = gpu.rand(784,1000);
  //w2 = gpu.rand(1000,10);

  printf("Finished!");
}

void MPI_benchmark(int argc, char *argv[])
{
    int myrank;
    MPI_Status status;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myrank);

    ClusterNet gpu = ClusterNet();
    int batch_rows = 128;
    int w_in = 10000;
    int w_out = 8000;

    //dot
    Matrix B = gpu.rand(w_in,w_out);
    Matrix A = gpu.rand(batch_rows,w_in);
    Matrix out = empty(batch_rows, w_out);

    Matrix B1 = gpu.rand(w_in,w_out/2);
    Matrix B2 = gpu.rand(w_in,w_out/2);
    Matrix D = empty(batch_rows,w_out/2);
    Matrix A1 = gpu.rand(batch_rows/2,w_in);
    Matrix big_out = gpu.rand(batch_rows/2,w_out);
    Matrix grand_out = empty(batch_rows, w_out);

    Matrix C = gpu.rand(batch_rows/2,w_in);
    Matrix C_out = empty(batch_rows/2,w_out);

    Matrix E = gpu.rand(batch_rows/4,w_in);
    Matrix E_out = empty(batch_rows/4,w_out);
    Matrix E_merge = empty(batch_rows/2,w_out);
    Matrix E_merge2 = empty(batch_rows/2,w_out);

    //add
	/*
    B = gpu.rand(w_in,w_out);
    A = gpu.rand(w_in,w_out);
    out = empty(w_in, w_out);
    A1 = gpu.rand(gen,w_in/2,w_out);
    A2 = gpu.rand(gen,w_in/2,w_out);
    D = empty(w_in/2,w_out);



    hipEvent_t* startstop = tick();
    for(int i = 0; i< 100; i++)
    {
      gpu.dot(A,B, out);
	//add(A, B, out);
    }
    printf("Direct compute:\n");
    tock(startstop);


    out = empty(batch_rows,w_out/2);
    Matrix out2 = empty(batch_rows,w_out/2);
    startstop = tick();
    for(int i = 0; i< 100; i++)
    {
      gpu.dot(A,B1, out);
      gpu.dot(A,B2, out2);
      merge(out,out2,grand_out);
    }
    printf("Direct compute x2:\n");
    tock(startstop);

    Matrix mergemat = empty(batch_rows, w_out);
    out = empty(batch_rows,w_out/2);
    startstop = tick();
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
	    if(myrank == 0)
	    {
		gpu.dot(A,B1, out);
    		//add(A1, B,out);
		MPI_Send(out.data, out.size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
	    }
	    else
	    {
		gpu.dot(A,B2, out);
		//add(A2,B, out);
	 	MPI_Recv(D.data, D.size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
                merge(out,D, mergemat);
	    }

    }

    if(myrank == 1)
    {
      printf("GPUDirect RDMA:\n");
      tock(startstop);
    }

    out = empty(batch_rows/2,w_out);
    startstop = tick();
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
	    if(myrank == 0)
	    {
		gpu.dot(C,B, out);
    		//add(A1, B,out);
		MPI_Send(out.data, out.size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
	    }
	    else
	    {
		gpu.dot(C,B, out);
		//add(A2,B, out);
	 	MPI_Recv(C_out.data, C_out.size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
                merge(out,C_out, grand_out);
	    }

    }

    if(myrank == 1)
    {
      printf("GPUDirect RDMA batch:\n");
      tock(startstop);
    }
/*

    out = empty(batch_rows/2,w_out);
    startstop = tick();
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
	    if(myrank == 0)
	    {
		gpu.dot(C,B, out);

	    }
	    else
	    {
		gpu.dot(C,B, out);
	    }

    }

    if(myrank == 1)
    {
      printf("GPUDirect RDMA batch compute:\n");
      tock(startstop);
    }


    out = empty(batch_rows/2,w_out);
    startstop = tick();
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
	    if(myrank == 0)
	    {
		MPI_Send(out.data, out.size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
	    }
	    else
	    {
	 	MPI_Recv(C_out.data, C_out.size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
	    }

    }

    if(myrank == 1)
    {
      printf("GPUDirect RDMA send:\n");
      tock(startstop);
    }


    out = empty(batch_rows/4,w_out);
    startstop = tick();
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
	    if(myrank == 0)
	    {
		gpu.dot(E,B, out);
		gpu.dot(E,B, E_out);
                merge(out,E_out, E_merge);
		MPI_Send(E_merge.data, E_merge.size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
	    }
	    else
	    {
		gpu.dot(E,B, out);
		gpu.dot(E,B, E_out);
                merge(out,E_out, E_merge);
	 	MPI_Recv(E_merge2.data, E_merge2.size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
                merge(E_merge2,E_merge, grand_out);
	    }

    }

    if(myrank == 1)
    {
      printf("GPUDirect RDMA batch x2:\n");
      tock(startstop);
    }



    out = empty(batch_rows/2,w_out/2);
    startstop = tick();
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
	    if(myrank == 0)
	    {
		gpu.dot(A1,B1, out);
		gpu.dot(A1,B1, out);
    		//add(A1, B,out);
                merge(A1,B1,big_out);
		MPI_Send(big_out.data, big_out.size, MPI_FLOAT, 1, 101, MPI_COMM_WORLD);
	    }
	    else
	    {
		gpu.dot(A1,B2, out);
		gpu.dot(A1,B2, out);
		//add(A2,B, out);
                merge(A1,B2,big_out);
	 	MPI_Recv(big_out.data, big_out.size, MPI_FLOAT, 0, 101, MPI_COMM_WORLD, &status);
	    }

    }

    if(myrank == 1)
    {
      printf("GPUDirect RDMA x2:\n");
      tock(startstop);
    }
*/


    MPI_Finalize();

}







int main(int argc, char *argv[])
{

  //MPI_benchmark(argc, argv);
	//ClusterNet *gpu new ClusterNet();


	ClusterNet gpu = ClusterNet();

	Matrix A = ones(10,10);
	Matrix B = ones(10,10);

	Matrix C = gpu.rand(2,2);
	Matrix D = gpu.randn(2,2);

	print_gpu_matrix(C);
	print_gpu_matrix(D);

	Matrix E = gpu.dot(C,D);

	print_gpu_matrix(E);




}


