#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include <util.cuh>
#include <basicOps.cuh>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <util.cuh>
#include <clusterNet.cuh>
#include <time.h>

void run_neural_network()
{
  Matrix X = read_csv("/home/tim/Downloads/mnist_full_X.csv");
  Matrix y = read_csv("/home/tim/Downloads/mnist_full_y.csv");
  printf("read MNIST\n");

  ClusterNet gpu = ClusterNet();

  Matrix result;
  Matrix w1 = gpu.rand(784,1000);
  Matrix w2 = gpu.rand(1000,10);

  printf("init batch allocator\n");
  gpu.init_batch_allocator(X, y, 124);

  clock_t t1,t2;
  t1=clock();
  //code goes here

  gpu.tick();
  for(int i = 0; i < gpu.m_total_batches; i++)
  {
	  gpu.allocate_next_batch_async();


	  result = gpu.dot(gpu.m_current_batch_X,w1);
	  result = gpuExp(result);
	  result = gpu.dot(result,w2);

	  gpu.replace_current_batch_with_next();

  }
  hipDeviceSynchronize();
  t2=clock();
  float diff ((float)t2-(float)t1);
  float mseconds = (diff / CLOCKS_PER_SEC)/1000;
  std::cout<<mseconds<<std::endl;
  gpu.tock();

  gpu.finish_batch_allocator();
  //gpu.tock("batch replace");
  //gpu.tock("async batch allocate");
  //gpu.tock("feedforward");


  printf("Finished!\n");
}

void MPI_benchmark(int argc, char *argv[])
{
    int myrank;
    MPI_Status status;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myrank);

    ClusterNet gpu = ClusterNet();
    int batch_rows = 128;
    int w_in = 10000;
    int w_out = 8000;

    //dot
    Matrix B = gpu.rand(w_in,w_out);
    Matrix A = gpu.rand(batch_rows,w_in);
    assert(test_matrix(A,batch_rows,w_in));
    assert(test_matrix(B,w_in,w_out));
    Matrix out = empty(batch_rows, w_out);

    Matrix B1 = gpu.rand(w_in,w_out/2);
    Matrix B2 = gpu.rand(w_in,w_out/2);
    Matrix D = empty(batch_rows,w_out/2);
    Matrix A1 = gpu.rand(batch_rows/2,w_in);
    Matrix big_out = gpu.rand(batch_rows/2,w_out);
    Matrix grand_out = empty(batch_rows, w_out);

    Matrix C = gpu.rand(batch_rows/2,w_in);
    Matrix C_out = empty(batch_rows/2,w_out);

    Matrix E = gpu.rand(batch_rows/4,w_in);
    Matrix E_out = empty(batch_rows/4,w_out);
    Matrix E_merge = empty(batch_rows/2,w_out);
    Matrix E_merge2 = empty(batch_rows/2,w_out);

    //add

    /*
    B = gpu.rand(w_in,w_out);
    A = gpu.rand(w_in,w_out);
    out = empty(w_in, w_out);
    A1 = gpu.rand(w_in/2,w_out);
    Matrix A2 = gpu.rand(w_in/2,w_out);
    D = empty(w_in/2,w_out);
*/

    hipEvent_t* startstop = tick();
    for(int i = 0; i< 100; i++)
    {
      gpu.dot(A,B, out);
	//add(A, B, out);
    }
    printf("Direct compute:\n");
    tock(startstop);


    out = empty(batch_rows,w_out/2);
    Matrix out2 = empty(batch_rows,w_out/2);
    startstop = tick();
    for(int i = 0; i< 100; i++)
    {
      gpu.dot(A,B1, out);
      gpu.dot(A,B2, out2);
      vStack(out,out2,grand_out);
    }
    printf("Direct compute x2:\n");
    tock(startstop);

    Matrix mergemat = empty(batch_rows, w_out);
    out = empty(batch_rows,w_out/2);
    startstop = tick();
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
	    if(myrank == 0)
	    {
		gpu.dot(A,B1, out);
    		//add(A1, B,out);
		MPI_Send(out.data, out.size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
	    }
	    else
	    {
		gpu.dot(A,B2, out);
		//add(A2,B, out);
	 	MPI_Recv(D.data, D.size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
                vStack(out,D, mergemat);
	    }

    }

    if(myrank == 1)
    {
      printf("GPUDirect RDMA:\n");
      tock(startstop);
    }

    out = empty(batch_rows/2,w_out);
    startstop = tick();
    gpu.tick("aa");
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
    	gpu.tick("dot");
		gpu.dot(C,B, out);
		gpu.tick("dot");

	    if(myrank == 0)
	    {
    		//add(A1, B,out);
		gpu.tick("send");
		MPI_Send(out.data, out.size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
		gpu.tick("send");
	    }
	    else
	    {
		//add(A2,B, out);
		gpu.tick("receive");
	 	MPI_Recv(C_out.data, C_out.size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
                vStack(out,C_out, grand_out);
                gpu.tick("receive");
	    }

	    if(myrank == 1)
	    {
    		//add(A1, B,out);
		gpu.tick("send");
		MPI_Send(out.data, out.size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD);
		gpu.tick("send");
	    }
	    else
	    {
		//add(A2,B, out);
		gpu.tick("receive");
	 	MPI_Recv(C_out.data, C_out.size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD, &status);
                vStack(out,C_out, grand_out);
                gpu.tick("receive");
	    }

    }

    gpu.tock("dot");

    if(myrank == 1)
    {
      printf("GPUDirect RDMA batch:\n");
      tock(startstop);

      gpu.tock("receive");
      gpu.tock("aa");
    }
    else
    {

        gpu.tock("send");
    }










    MPI_Finalize();

}



void dotMPI_test(int argc, char *argv[])
{
	ClusterNet gpu = ClusterNet(argc, argv, 123465);
	Matrix A = gpu.rand(128,1000);
	Matrix B = gpu.rand(1000,400);

	gpu.tick("dot mpi batch");
	for(int i = 0; i < 100; i++)
	{
		gpu.dotMPI_batchSlice(A,B);
	}
	gpu.tock("dot mpi batch");



	gpu.tick("dot mpi unit");
	for(int i = 0; i < 100; i++)
	{
		gpu.dotMPI_unitSlice(A,B);
	}
	gpu.tock("dot mpi unit");

	printf("My rank: %i\n",gpu.m_rank);
	gpu.benchmark_dot();



	gpu.tick("dot normal");
	for(int i = 0; i < 100; i++)
	{
		gpu.dot(A,B);
	}
	gpu.tock("dot normal");



	gpu.shutdown_MPI();
}



int main(int argc, char *argv[])
{

  //MPI_benchmark(argc, argv);


	run_neural_network();





}



