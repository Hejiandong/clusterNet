#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include <util.cuh>
#include <basicOps.cuh>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <util.cuh>
#include <clusterNet.h>
#include <time.h>
#include <batchAllocator.h>
#include <DeepNeuralNetwork.h>
#include <WikiMaxoutNet.h>
#include <WikiMaxoutNet_PCIe.h>
#include <WikiMaxoutNet_PCIe2.h>
#include <WikiNetDist.h>
#include <Layer.h>
#include <time.h>

using std::cout;
using std::endl;





void run_neural_network()
{
  Matrix *X = read_hdf5("/home/tim/mnist_full_X.hdf5");
  Matrix *y = read_hdf5("/home/tim/mnist_full_y.hdf5");

  ClusterNet gpu = ClusterNet(12345);

  cout << X->rows << endl;

  int hidden_size = 1024;


  Matrix *w1 = gpu.sparseInitWeight(784,hidden_size);
  Matrix *w2 = gpu.sparseInitWeight(hidden_size,10);
  Matrix *m1 = zeros(784,hidden_size);
  Matrix *m2 = zeros(hidden_size,10);
  Matrix *ms1 = zeros(784,hidden_size);
  Matrix *ms2 = zeros(hidden_size,10);
  Matrix *grad_w1_ms = zeros(784,hidden_size);
  Matrix *grad_w2_ms = zeros(hidden_size,10);
  Matrix *grad_w2 = empty(hidden_size,10);
  Matrix *grad_w1 = empty(784,hidden_size);
  float cv_error = 0;
  float cv_size = 0.1428571f;
  float train_error = 0.0f;

  BatchAllocator b = BatchAllocator();
  b.init(X, y,  cv_size, 128, 512);

  clock_t t1,t2;
  t1=clock();
  //code goes here
  int epochs  = 100;
  gpu.tick();
  float learning_rate = 0.003;
  //size_t free = 0;
  //size_t total = 0;
  float momentum = 0.5;
  for(int EPOCH = 0; EPOCH < epochs; EPOCH++)
  {
	  std::cout << "EPOCH: " << EPOCH + 1 << std::endl;
	  //hipMemGetInfo(&free, &total);
	  //std::cout << free << std::endl;
	  momentum += 0.01;
	  if(momentum > 0.95) momentum = 0.95;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {

		  b.allocate_next_batch_async();

		  //nesterov updates
		  scalarMul(m1,momentum,m1);
		  scalarMul(m2,momentum,m2);
		  add(w1,m1,w1);
		  add(w2,m2,w2);

		  Matrix *d0 = gpu.dropout(b.CURRENT_BATCH,0.2);
		  Matrix *z1 = gpu.dot(d0, w1);
		  logistic(z1, z1);
		  Matrix *d1 = gpu.dropout(z1,0.5);
		  Matrix *a2 = gpu.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *t = create_t_matrix(b.CURRENT_BATCH_Y,10);

		  //backprop
		  Matrix *e1 = sub(out, t);
		  Matrix *e2 = gpu.dotT(e1, w2);
		  gpu.Tdot(z1,e1,grad_w2);
		  logisticGrad(z1,z1);
		  mul(e2,z1,e2);
		  gpu.Tdot(b.CURRENT_BATCH,e2,grad_w1);

		  b.allocate_next_batch_async();

		  RMSprop_with_momentum_weight_update(ms1,grad_w1,w1,m1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_momentum_weight_update(ms2,grad_w2,w2,m2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);

		  hipFree(e1->data);
		  hipFree(e2->data);
		  hipFree(z1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(t->data);
		  hipFree(d0->data);
		  hipFree(d1->data);

		  b.replace_current_batch_with_next();

	  }


	  //Matrix *sum_value = sum(w1);
	  //std::cout << "weight 1 Sum: " << to_host(sum_value)->data[0] << std::endl;

	  train_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {

		  b.broadcast_batch_to_processes();

		  //Matrix *d0 = scalarMul(b.CURRENT_BATCH,0.8);
		  Matrix *a1 = gpu.dot(b.CURRENT_BATCH,w1);
		  logistic(a1, a1);
		  //Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpu.dot(a1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_Y);
		  b.allocate_next_batch_async();
		  float sum_value = sum(eq);

		  train_error += (b.CURRENT_BATCH->rows - sum_value)/ (1.0f * b.CURRENT_BATCH->rows *b.TOTAL_BATCHES) ;

		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);
		  //hipFree(d0->data);
		  //hipFree(d1->data);

		  b.replace_current_batch_with_next();
	  }


	  std::cout << "Train error: " << train_error << std::endl;


	  cv_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES_CV; i++)
	  {
		  b.broadcast_batch_cv_to_processes();
		  Matrix *d0 = scalarMul(b.CURRENT_BATCH_CV,0.8);
		  Matrix *a1 = gpu.dot(d0,w1);
		  logistic(a1, a1);
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpu.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_CV_Y);
		  b.allocate_next_cv_batch_async();
		  float sum_value = sum(eq);

		  cv_error += (b.CURRENT_BATCH_CV->rows  - sum_value)/ (1.0f * b.CURRENT_BATCH_CV->rows *b.TOTAL_BATCHES_CV) ;

		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);
		  hipFree(d0->data);
		  hipFree(d1->data);

		  b.replace_current_cv_batch_with_next();
	  }

	  std::cout << "Cross validation error: " << cv_error << std::endl;


  }

  hipDeviceSynchronize();
  t2=clock();
  float diff ((float)t2-(float)t1);
  float mseconds = (diff / CLOCKS_PER_SEC)/1000;
  std::cout<<mseconds<<std::endl;
  gpu.tock();

  b.finish_batch_allocator();

  //gpu.tock("batch replace");
  //gpu.tock("async batch allocate");
  //gpu.tock("feedforward");


  printf("Finished!\n");
}


void run_maxout_network()
{

	hipSetDevice(0);
    Matrix *X = read_hdf5("/home/tim/mnist_full_X.hdf5");
    Matrix *y = read_hdf5("/home/tim/mnist_full_y.hdf5");

  	ClusterNet gpus = ClusterNet(12345);

  	int hiddenunits = 512;
  	int maxout_Size = 8;
  	int batch_size = 128;

	Matrix *w1 = gpus.uniformSqrtWeight(784,hiddenunits);
	Matrix *w2 = gpus.uniformSqrtWeight(hiddenunits/maxout_Size,10);
	Matrix *b1 = zeros(1,hiddenunits);
	Matrix *b2 = zeros(1,10);
	Matrix *m1 = zeros(784,hiddenunits);
	Matrix *m2 = zeros(hiddenunits/maxout_Size,10);
	Matrix *mb1 = zeros(1,hiddenunits);
	Matrix *mb2 = zeros(1,10);
	Matrix *ms1 = zeros(784,hiddenunits);
	Matrix *ms2 = zeros(hiddenunits/maxout_Size,10);
	Matrix *msb1 = zeros(1,hiddenunits);
	Matrix *msb2 = zeros(1,10);
	Matrix *grad_w1 = zeros(784,hiddenunits);
	Matrix *grad_w2 = zeros(hiddenunits/maxout_Size,10);
	Matrix *grad_b1 = zeros(1,hiddenunits);
	Matrix *grad_b2 = zeros(1,10);


	float cv_error = 0.0f;
	float train_error = 0.0f;

	BatchAllocator b = BatchAllocator();
	b.init(X, y, 0.2, batch_size, 512);
	int epochs  = 1000;
	float learning_rate = 0.001;
	float momentum = 0.5;
	for(int EPOCH = 1; EPOCH < epochs; EPOCH++)
	{
	  cout << "EPOCH: " << EPOCH << endl;
	  //momentum += 0.01;
	  //if(momentum > 0.95) momentum = 0.95;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {
		  b.broadcast_batch_to_processes();

		  //nesterov updates
		  scalarMul(m1,momentum,m1);
		  scalarMul(m2,momentum,m2);
		  scalarMul(mb1,momentum,mb1);
		  scalarMul(mb2,momentum,mb2);
		  add(w1,m1,w1);
		  add(w2,m2,w2);
		  add(b1,mb1,b1);
		  add(b2,mb2,b2);


		  //feedforward
		  Matrix *d0 = gpus.dropout(b.CURRENT_BATCH,0.2);
		  Matrix *z1 = gpus.dot(d0, w1);
		  addMatrixVector(z1,b1,z1);
		  Matrix **a_paired = maxout(z1,maxout_Size);
		  Matrix *a1 = a_paired[0];
		  Matrix *a1_idx = a_paired[1];
		  Matrix *d1 = gpus.dropout(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  addMatrixVector(a2,b2,a2);
		  Matrix *out = softmax(a2);
		  Matrix *t = create_t_matrix(b.CURRENT_BATCH_Y,10);

		  b.allocate_next_batch_async();

		  //backprop
		  Matrix *e1 = sub(out, t);
		  Matrix *e2_partial = gpus.dotT(e1, w2);
		  Matrix *e2 = empty(b.CURRENT_BATCH->rows,e2_partial->cols*maxout_Size);
		  Matrix *aB = ones(1,b.CURRENT_BATCH->rows);


		  gpus.Tdot(a1,e1,grad_w2);
		  gpus.dot(aB,e1,grad_b2);
		  expand_to_maxout_grad(e2_partial, a1_idx,e2);
		  gpus.Tdot(b.CURRENT_BATCH,e2,grad_w1);
		  gpus.dot(aB,e2,grad_b1);

		  //weight updates
		  //RMSProp


		  RMSprop_with_momentum_weight_update(ms1,grad_w1,w1,m1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_momentum_weight_update(ms2,grad_w2,w2,m2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);

		  RMSprop_with_momentum_weight_update(msb1,grad_b1,b1,mb1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_momentum_weight_update(msb2,grad_b2,b2,mb2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);



/*
		  scalarMul(grad_w1,learning_rate/(float)b.CURRENT_BATCH->rows,grad_w1);
		  scalarMul(grad_w2,learning_rate/(float)b.CURRENT_BATCH->rows,grad_w2);
		  scalarMul(grad_b1,learning_rate/(float)b.CURRENT_BATCH->rows,grad_b1);
		  scalarMul(grad_b2,learning_rate/(float)b.CURRENT_BATCH->rows,grad_b2);



		  //classical momentum
		  scalarMul(m1,momentum,m1);
		  scalarMul(m2,momentum,m2);
		  scalarMul(mb1,momentum,mb1);
		  scalarMul(mb2,momentum,mb2);
		  sub(m1,grad_w1,m1);
		  sub(m2,grad_w2,m2);
		  sub(mb1,grad_b1,mb1);
		  sub(mb2,grad_b2,mb2);

		  add(w1,m1,w1);
		  add(w2,m2,w2);
		  add(b1,mb1,b1);
		  add(b2,mb2,b2);

		  */



		  /*
		  sub(w1,grad_w1,w1);
		  sub(w2,grad_w2,w2);
		  sub(b1,grad_b1,b1);
		  sub(b2,grad_b2,b2);
		  */



		  hipFree(e1->data);
		  hipFree(e2->data);
		  hipFree(e2_partial->data);
		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a1_idx->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(t->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  hipFree(aB->data);
		  free(a_paired);

		  b.replace_current_batch_with_next();

	  }



	  train_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {

		  b.broadcast_batch_to_processes();

		  Matrix *d0 = scalarMul(b.CURRENT_BATCH,0.8);
		  Matrix *z1 = gpus.dot(d0,w1);
		  Matrix **a1_pair = maxout(z1,maxout_Size);
		  Matrix *a1 = a1_pair[0];
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_Y);
		  b.allocate_next_batch_async();
		  float sum_value = sum(eq);

		  train_error += (b.CURRENT_BATCH->rows - sum_value)/ (1.0f * b.CURRENT_BATCH->rows *b.TOTAL_BATCHES) ;

		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a1_pair[1]->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  free(a1_pair);

		  b.replace_current_batch_with_next();
	  }

	  std::cout << "MAXOUT Train error: " << train_error << std::endl;



	  cv_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES_CV; i++)
	  {
		  b.broadcast_batch_cv_to_processes();
		  Matrix *d0 = scalarMul(b.CURRENT_BATCH_CV,0.8);
		  Matrix *z1 = gpus.dot(d0,w1);
		  Matrix **a1_pair = maxout(z1,maxout_Size);
		  Matrix *a1 = a1_pair[0];
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_CV_Y);
		  b.allocate_next_batch_async();
		  float sum_value = sum(eq);

		  cv_error += (b.CURRENT_BATCH_CV->rows  - sum_value)/ (1.0f * b.CURRENT_BATCH_CV->rows *b.TOTAL_BATCHES_CV) ;

		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a1_pair[1]->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  free(a1_pair);

		  b.replace_current_cv_batch_with_next();
	  }

	  std::cout << "MAXOUT Cross validation error: " << cv_error << std::endl;

	}

}


void run_normal_net()
{
	hipSetDevice(2);
    Matrix *X = read_hdf5("/home/tim/mnist_full_X.hdf5");
    Matrix *y = read_hdf5("/home/tim/mnist_full_y.hdf5");

  	ClusterNet gpus = ClusterNet(12345);

  	int hiddenunits = 1024;
  	int maxout_Size = 1;
  	int batch_size = 128;

	Matrix *w1 = gpus.uniformSqrtWeight(784,hiddenunits);
	Matrix *w2 = gpus.uniformSqrtWeight(hiddenunits/maxout_Size,10);
	Matrix *b1 = zeros(1,hiddenunits);
	Matrix *b2 = zeros(1,10);
	Matrix *m1 = zeros(784,hiddenunits);
	Matrix *m2 = zeros(hiddenunits/maxout_Size,10);
	Matrix *mb1 = zeros(1,hiddenunits);
	Matrix *mb2 = zeros(1,10);
	Matrix *ms1 = zeros(784,hiddenunits);
	Matrix *ms2 = zeros(hiddenunits/maxout_Size,10);
	Matrix *msb1 = zeros(1,hiddenunits);
	Matrix *msb2 = zeros(1,10);
	Matrix *grad_w1 = zeros(784,hiddenunits);
	Matrix *grad_w2 = zeros(hiddenunits/maxout_Size,10);
	Matrix *grad_b1 = zeros(1,hiddenunits);
	Matrix *grad_b2 = zeros(1,10);


	float cv_error = 0.0f;
	float train_error = 0.0f;

	BatchAllocator b = BatchAllocator();
	b.init(X, y, 0.4, batch_size, 512);
	int epochs  = 500;
	float learning_rate = 0.000001;
	float momentum = 0.5;
	for(int EPOCH = 1; EPOCH < epochs; EPOCH++)
	{
	  cout << "EPOCH: " << EPOCH << endl;
	  momentum += 0.01;
	  if(momentum > 0.95) momentum = 0.95;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {
		  b.broadcast_batch_to_processes();

		  //nesterov updates
		  scalarMul(m1,momentum,m1);
		  scalarMul(m2,momentum,m2);
		  scalarMul(mb1,momentum,mb1);
		  scalarMul(mb2,momentum,mb2);
		  add(w1,m1,w1);
		  add(w2,m2,w2);
		  add(b1,mb1,b1);
		  add(b2,mb2,b2);







		  //feedforward
		  Matrix *d0 = gpus.dropout(b.CURRENT_BATCH,0.2);
		  Matrix *z1 = gpus.dot(d0, w1);
		  addMatrixVector(z1,b1,z1);
		  Matrix *a1 = logistic(z1);
		  //Matrix *a1 = rectified_linear(z1);
		  Matrix *d1 = gpus.dropout(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  addMatrixVector(a2,b2,a2);
		  Matrix *out = softmax(a2);
		  Matrix *t = create_t_matrix(b.CURRENT_BATCH_Y,10);

		  b.allocate_next_batch_async();

		  //backprop
		  Matrix *e1 = sub(out, t);
		  Matrix *e2 = gpus.dotT(e1, w2);
		  Matrix *aB = ones(1,b.CURRENT_BATCH->rows);


		  gpus.Tdot(a1,e1,grad_w2);
		  gpus.dot(aB,e1,grad_b2);
		  //rectified_linear_derivative(a1,a1);
		  logisticGrad(a1,a1);
		  mul(e2,a1,e2);
		  gpus.Tdot(b.CURRENT_BATCH,e2,grad_w1);
		  gpus.dot(aB,e2,grad_b1);



		  /*
		  //about equal to momentum update + nesterov update -> momentum applyied to gradient+momentum better?
		  RMSprop_with_momentum_weight_update(ms1,grad_w1,w1,m1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_momentum_weight_update(ms2,grad_w2,w2,m2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);

		  RMSprop_with_momentum_weight_update(msb1,grad_b1,b1,mb1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_momentum_weight_update(msb2,grad_b2,b2,mb2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  */

		  /*
		  //slow and generally worse error, but sometimes better results in the end
		  RMSprop_with_momentum_update(ms1,grad_w1,w1,m1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_momentum_update(ms2,grad_w2,w2,m2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);

		  RMSprop_with_momentum_update(msb1,grad_b1,b1,mb1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_momentum_update(msb2,grad_b2,b2,mb2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  */




		  RMSprop_with_nesterov_weight_update(ms1,grad_w1,w1,m1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_nesterov_weight_update(ms2,grad_w2,w2,m2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);

		  RMSprop_with_nesterov_weight_update(msb1,grad_b1,b1,mb1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_nesterov_weight_update(msb2,grad_b2,b2,mb2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);


		  /*
		  //slower but equally good to nesterov momentum
		  RMSprop_with_weight_update(ms1,grad_w1,w1,m1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_weight_update(ms2,grad_w2,w2,m2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);

		  RMSprop_with_weight_update(msb1,grad_b1,b1,mb1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_weight_update(msb2,grad_b2,b2,mb2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  */
		  /*





		  scalarMul(grad_w1,learning_rate/(float)b.CURRENT_BATCH->rows,grad_w1);
		  scalarMul(grad_w2,learning_rate/(float)b.CURRENT_BATCH->rows,grad_w2);
		  scalarMul(grad_b1,learning_rate/(float)b.CURRENT_BATCH->rows,grad_b1);
		  scalarMul(grad_b2,learning_rate/(float)b.CURRENT_BATCH->rows,grad_b2);



		  //classical momentum
		  scalarMul(m1,momentum,m1);
		  scalarMul(m2,momentum,m2);
		  scalarMul(mb1,momentum,mb1);
		  scalarMul(mb2,momentum,mb2);
		  sub(m1,grad_w1,m1);
		  sub(m2,grad_w2,m2);
		  sub(mb1,grad_b1,mb1);
		  sub(mb2,grad_b2,mb2);


		  add(w1,m1,w1);
		  add(w2,m2,w2);
		  add(b1,mb1,b1);
		  add(b2,mb2,b2);
		  */




		  /*
		  sub(w1,grad_w1,w1);
		  sub(w2,grad_w2,w2);
		  sub(b1,grad_b1,b1);
		  sub(b2,grad_b2,b2);
		  */



		  hipFree(e1->data);
		  hipFree(e2->data);
		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(t->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  hipFree(aB->data);

		  b.replace_current_batch_with_next();

	  }



	  train_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {

		  b.broadcast_batch_to_processes();

		  Matrix *d0 = scalarMul(b.CURRENT_BATCH,0.8);
		  Matrix *z1 = gpus.dot(d0,w1);
		  Matrix *a1 = logistic(z1);
		  //Matrix *a1 = rectified_linear(z1);
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_Y);
		  b.allocate_next_batch_async();
		  float sum_value = sum(eq);

		  train_error += (b.CURRENT_BATCH->rows - sum_value)/ (1.0f * b.CURRENT_BATCH->rows *b.TOTAL_BATCHES) ;

		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);
		  hipFree(d0->data);
		  hipFree(d1->data);

		  b.replace_current_batch_with_next();
	  }

	  std::cout << "MAXOUT Train error: " << train_error << std::endl;



	  cv_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES_CV; i++)
	  {
		  b.broadcast_batch_cv_to_processes();
		  Matrix *d0 = scalarMul(b.CURRENT_BATCH_CV,0.8);
		  Matrix *z1 = gpus.dot(d0,w1);
		  Matrix *a1 = logistic(z1);
		  //Matrix *a1 = rectified_linear(z1);
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_CV_Y);
		  b.allocate_next_batch_async();
		  float sum_value = sum(eq);

		  cv_error += (b.CURRENT_BATCH_CV->rows  - sum_value)/ (1.0f * b.CURRENT_BATCH_CV->rows *b.TOTAL_BATCHES_CV) ;

		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);
		  hipFree(d0->data);
		  hipFree(d1->data);

		  b.replace_current_cv_batch_with_next();
	  }

	  std::cout << "MAXOUT Cross validation error: " << cv_error << std::endl;

	}

}

void MPI_benchmark_P2P(int argc, char *argv[])
{
	char name[100];
    int myrank, length, size;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
    MPI_Get_processor_name(name, &length);
	MPI_Comm_size(MPI_COMM_WORLD, &size);
	MPI_Status status;

	int local_rank = myrank % 4;

	int gpus;
	hipGetDeviceCount(&gpus);
	int mygpu_id;
	int your_gpu_id;
	if(myrank == 0)
	{
		mygpu_id = 0;
		if(gpus > 1)
			your_gpu_id = 1;
		else
			your_gpu_id = 0;

		MPI_Send(&your_gpu_id,1, MPI_INT,1,0,MPI_COMM_WORLD);
	}
	else
	{
		MPI_Recv(&mygpu_id,1,MPI_INT,myrank-1,0,MPI_COMM_WORLD,&status);
		if(gpus > mygpu_id+1)
			your_gpu_id = mygpu_id + 1;
		else
			your_gpu_id = 0;
		if(myrank < size-1)
			MPI_Send(&your_gpu_id,1, MPI_INT,myrank+1,0,MPI_COMM_WORLD);
	}

	hipSetDevice(mygpu_id);


		int batch_size = 128;
		int inner_dim = 10000;
		int outer_dim = 15000;

		ClusterNet gpu = ClusterNet();
		Matrix *A = gpu.rand(batch_size,inner_dim);
		Matrix *B = gpu.rand(inner_dim,outer_dim);
		Matrix *out = empty(batch_size,outer_dim);
		Matrix *rec = empty(batch_size,outer_dim);

		Matrix *A1 = gpu.rand(batch_size/2,inner_dim);
		Matrix *B1 = gpu.rand(inner_dim,outer_dim);
		Matrix *rec1 = empty(batch_size/2,outer_dim);
		Matrix *out1 = empty(batch_size/2,outer_dim);

		Matrix *A2 = gpu.rand(batch_size,inner_dim);
		Matrix *B2 = gpu.rand(inner_dim,outer_dim/2);
		Matrix *rec2 = empty(batch_size,outer_dim/2);
		Matrix *out2 = empty(batch_size,outer_dim/2);


		gpu.tick("Direct compute");
	    for(int i = 0; i< 100; i++)
	    {
	      gpu.dot(A,B, out);
		//add(A, B, out);
	    }
	    gpu.tock("Direct compute");

		gpu.tick("partial batch direct compute");
	    for(int i = 0; i< 100; i++)
	    {
	      gpu.dot(A1,B1, out1);
		//add(A, B, out);
	    }
	    gpu.tock("partial batch direct compute");

		gpu.tick("partial units direct compute");
	    for(int i = 0; i< 100; i++)
	    {
	      gpu.dot(A2,B2, out2);
		//add(A, B, out);
	    }
	    gpu.tock("partial units direct compute");




		gpu.tick("PCIe transfer");
		for(int i = 0; i< 100; i++)
		{
			if(local_rank == 0 && gpus > 1)
			{
				MPI_Send(out->data, out->size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
			}
			else if(local_rank == 1 && gpus > 1)
			{
				//add(A2,B, out);
				MPI_Recv(rec->data, rec->size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
			}
		}
		gpu.tock("PCIe transfer");


		gpu.tick("PCIe dot");
		for(int i = 0; i< 100; i++)
		{
			if(local_rank == 0 && gpus > 1)
			{
				gpu.dot(A2,B2,out2);
				MPI_Send(out1->data, out1->size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
			}
			else if(local_rank == 1 && gpus > 1)
			{
				gpu.dot(A2,B2,out2);
				MPI_Recv(rec1->data, rec1->size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
				vStack(out2,rec2,rec);
			}
		}
		gpu.tock("PCIe dot");



		gpu.tick("RDMA transfer");
		for(int i = 0; i< 100; i++)
		{
			if(myrank == 0)
			{
				MPI_Send(out->data, out->size, MPI_FLOAT, 3, 100, MPI_COMM_WORLD);
			}
			else if(myrank == 3)
			{
				//add(A2,B, out);
				MPI_Recv(rec->data, rec->size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
			}
		}
		gpu.tock("RDMA transfer");


		gpu.tick("RDMA dot");
		for(int i = 0; i< 100; i++)
		{
			if(myrank == 0)
			{
				gpu.dot(A2,B2,out2);
				MPI_Send(out->data, out->size, MPI_FLOAT, 3, 100, MPI_COMM_WORLD);
			}
			else if(myrank == 3)
			{
				//add(A2,B, out);
				gpu.dot(A2,B2,out2);
				MPI_Recv(rec->data, rec->size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
				vStack(out2,rec2,rec);
			}
		}
		gpu.tock("RDMA dot");








	MPI_Finalize();



}


void MPI_benchmark(int argc, char *argv[])
{
    int myrank;
    MPI_Status status;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myrank);

    ClusterNet gpu = ClusterNet();
    int batch_rows = 128;
    int w_in = 10000;
    int w_out = 8000;

    //dot
    Matrix *B = gpu.rand(w_in,w_out);
    Matrix *A = gpu.rand(batch_rows,w_in);
    assert(test_matrix(A,batch_rows,w_in));
    assert(test_matrix(B,w_in,w_out));
    Matrix *out = empty(batch_rows, w_out);

    Matrix *B1 = gpu.rand(w_in,w_out/2);
    Matrix *B2 = gpu.rand(w_in,w_out/2);
    Matrix *D = empty(batch_rows,w_out/2);
    Matrix *A1 = gpu.rand(batch_rows/2,w_in);
    Matrix *big_out = gpu.rand(batch_rows/2,w_out);
    Matrix *grand_out = empty(batch_rows, w_out);

    Matrix *C = gpu.rand(batch_rows/2,w_in);
    Matrix *C_out = empty(batch_rows/2,w_out);

    Matrix *E = gpu.rand(batch_rows/4,w_in);
    Matrix *E_out = empty(batch_rows/4,w_out);
    Matrix *E_merge = empty(batch_rows/2,w_out);
    Matrix *E_merge2 = empty(batch_rows/2,w_out);

    //add

    /*
    B = gpu.rand(w_in,w_out);
    A = gpu.rand(w_in,w_out);
    out = empty(w_in, w_out);
    A1 = gpu.rand(w_in/2,w_out);
    Matrix *A2 = gpu.rand(w_in/2,w_out);
    D = empty(w_in/2,w_out);
*/

    hipEvent_t* startstop = tick();
    for(int i = 0; i< 100; i++)
    {
      gpu.dot(A,B, out);
	//add(A, B, out);
    }
    printf("Direct compute:\n");
    tock(startstop);


    out = empty(batch_rows,w_out/2);
    Matrix *out2 = empty(batch_rows,w_out/2);
    startstop = tick();
    for(int i = 0; i< 100; i++)
    {
      gpu.dot(A,B1, out);
      gpu.dot(A,B2, out2);
      vStack(out,out2,grand_out);
    }
    printf("Direct compute x2:\n");
    tock(startstop);

    Matrix *mergemat = empty(batch_rows, w_out);
    out = empty(batch_rows,w_out/2);
    startstop = tick();
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
	    if(myrank == 0)
	    {
		gpu.dot(A,B1, out);
    		//add(A1, B,out);
		MPI_Send(out->data, out->size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
	    }
	    else
	    {
		gpu.dot(A,B2, out);
		//add(A2,B, out);
	 	MPI_Recv(D->data, D->size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
                vStack(out,D, mergemat);
	    }

    }

    if(myrank == 1)
    {
      printf("GPUDirect RDMA:\n");
      tock(startstop);
    }

    out = empty(batch_rows/2,w_out);
    startstop = tick();
    gpu.tick("aa");
    //out = empty(w_in/2,w_out);
    for(int i = 0; i < 100; i++)
    {
    	gpu.tick("dot");
		gpu.dot(C,B, out);
		gpu.tick("dot");

	    if(myrank == 0)
	    {
    		//add(A1, B,out);
		gpu.tick("send");
		MPI_Send(out->data, out->size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD);
		gpu.tick("send");
	    }
	    else
	    {
		//add(A2,B, out);
		gpu.tick("receive");
	 	MPI_Recv(C_out->data, C_out->size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &status);
                vStack(out,C_out, grand_out);
                gpu.tick("receive");
	    }

	    if(myrank == 1)
	    {
    		//add(A1, B,out);
		gpu.tick("send");
		MPI_Send(out->data, out->size, MPI_FLOAT, 0, 100, MPI_COMM_WORLD);
		gpu.tick("send");
	    }
	    else
	    {
		//add(A2,B, out);
		gpu.tick("receive");
	 	MPI_Recv(C_out->data, C_out->size, MPI_FLOAT, 1, 100, MPI_COMM_WORLD, &status);
                vStack(out,C_out, grand_out);
                gpu.tick("receive");
	    }

    }

    gpu.tock("dot");

    if(myrank == 1)
    {
      printf("GPUDirect RDMA batch:\n");
      tock(startstop);

      gpu.tock("receive");
      gpu.tock("aa");
    }
    else
    {

        gpu.tock("send");
    }










    MPI_Finalize();

}



void dotMPI_test(int argc, char *argv[])
{

	/*
	ClusterNet gpu = ClusterNet(argc, argv, 123465);
	int inner = 2000;
	int outer = 1200;
	int batch_size = 128;


	int reduced_left = 128;
	int reduced_right = 400;
	Matrix *A = gpu.rand(batch_size,inner);
	Matrix *B = gpu.rand(inner,outer);
	Matrix *A1 = gpu.rand(reduced_left,inner);
	Matrix *B1 = gpu.rand(inner,reduced_right);

	Matrix *out = empty(batch_size,outer);
	Matrix *out1 = empty(reduced_left,reduced_right);
	Matrix *recv1 = empty(reduced_left,reduced_right);
	Matrix *recv2 = empty(reduced_left,reduced_right);
	Matrix *recv3 = empty(reduced_left,reduced_right);
	MPI_Status status;




	gpu.tick("dot mpi batch");
	for(int i = 0; i < 100; i++)
	{
		gpu.dotMPI_batchSlice(A,B);
	}
	gpu.tock("dot mpi batch");



	gpu.tick("dot mpi unit");
	for(int i = 0; i < 100; i++)
	{
		gpu.dotMPI_unitSlice(A,B);
	}
	gpu.tock("dot mpi unit");

	printf("My rank: %i\n",gpu.MYRANK);
	//gpu.benchmark_dot();



	gpu.tick("dot normal");
	for(int i = 0; i < 100; i++)
	{
		gpu.dot(A,B,out);
	}
	gpu.tock("dot normal");



	//std::vector<MPI_Request> requests;
	MPI_Request *requests = (MPI_Request*)malloc(sizeof(MPI_Request)*gpu.MPI_SIZE-1);
	MPI_Request request_send;
	std::vector<Matrix*> recv_buffer;
	for(int i = 0; i < gpu.MPI_SIZE-1; i++)
	{
		MPI_Request request;
		requests[i] = request;
	}




	int received_count = 0;
	for(int i = 0; i < 100; i++)
	{
		for(int i = 0; i < recv_buffer.size(); i++)
			hipFree(recv_buffer[i]->data);
		recv_buffer.clear();
		out1 = empty(reduced_left,reduced_right);
		for(int i = 0; i < gpu.MPI_SIZE; i++)
		{
			recv_buffer.push_back(empty(reduced_left,reduced_right));
		}

		gpu.tick("all to all custom");
		//cout << "a1 rows" << A1->rows << endl;
		gpu.dot(A1,B1,out1);
		recv_buffer[gpu.MYRANK]= out1;
		for(int i = 0; i < gpu.MPI_SIZE; i++)
		{
			if(gpu.MYRANK == i) { continue; }
			MPI_Isend(out1->data, out1->size, MPI_FLOAT, i, 100, MPI_COMM_WORLD, &request_send);
		}

		for(int i = 0; i < gpu.MPI_SIZE; i++)
		{
			if(gpu.MYRANK == i) { continue; }
			MPI_Irecv(recv1->data, recv1->size, MPI_FLOAT, i, 100, MPI_COMM_WORLD, &requests[i]);

		}

		for(int i = 0; i < gpu.MPI_SIZE; i++)
		{
			if(gpu.MYRANK == i) { continue; }
			MPI_Wait(&requests[i],MPI_STATUS_IGNORE);
		}



		received_count = 0;
		while(received_count < gpu.MPI_SIZE-1)
		{
			for(int i = 0; i < gpu.MPI_SIZE; i++)
			{
				int received = 0;
				if(gpu.MYRANK == i) { continue; }
				MPI_Test(&requests[i],&received,&status);
				if(received == 1)
				{
					out1 = hStack(out1,recv1);
					received_count++;
				}
			}
		}

		gpu.tick("all to all custom");
	}
	gpu.tock("all to all custom");



	int destination = gpu.MYRANK + 1;
	int source = gpu.MYRANK - 1;
	if(destination == gpu.MPI_SIZE){destination = 0; }
	if(source < 0){ source = gpu.MPI_SIZE - 1;}
	for(int i = 0; i < 100; i++)
	{
		out1 = empty(reduced_left,reduced_right);
		recv1 = empty(reduced_left,reduced_right);
		gpu.tick("chain custom");
		gpu.dot(A1,B1,out1);
		for(int i = 0; i < gpu.MPI_SIZE-1; i++)
		{
			if(i == 0)
				MPI_Isend(out1->data, out1->size, MPI_FLOAT, destination, 100, MPI_COMM_WORLD, &request_send);
			else
				MPI_Isend(recv1->data, recv1->size, MPI_FLOAT, destination, 100, MPI_COMM_WORLD, &request_send);

			MPI_Recv(recv1->data, recv1->size, MPI_FLOAT, source, 100, MPI_COMM_WORLD, &status);

			//MPI_Wait(&requests[i],&status);
			out1 = hStack(out1,recv1);
		}
		gpu.tick("chain custom");
	}
	gpu.tock("chain custom");



	cout << gpu.MYRANK << endl;




	int matrix_idx = gpu.MYRANK;
	Matrix** arrOut = (Matrix**)malloc(sizeof(Matrix*)*gpu.MPI_SIZE);
	for(int i = 0; i < gpu.MPI_SIZE; i++)
		arrOut[i] = empty(reduced_left,reduced_right);

	float **h_arrA = (float**)malloc(sizeof(float*)*gpu.MPI_SIZE);
		for(int i = 0; i < gpu.MPI_SIZE; i++)
			h_arrA[i] = arrOut[i]->data;

	float **d_arrA;
	hipMalloc((void**) &d_arrA,sizeof(float*)*gpu.MPI_SIZE);
	hipMemcpy(d_arrA,h_arrA,sizeof(float*)*gpu.MPI_SIZE,hipMemcpyDefault);

	gpu.tick("chain matrix array");
	for(int i = 0; i < 100; i++)
	{
		gpu.dot(A1,B1,arrOut[gpu.MYRANK]);
		matrix_idx = gpu.MYRANK;
		for(int i = 0; i < gpu.MPI_SIZE-1; i++)
		{
			MPI_Isend(arrOut[matrix_idx]->data, arrOut[matrix_idx]->size, MPI_FLOAT, destination, 100, MPI_COMM_WORLD, &request_send);
			matrix_idx = (matrix_idx - 1) < 0 ? gpu.MPI_SIZE-1 : (matrix_idx - 1);
			MPI_Irecv(arrOut[matrix_idx]->data, arrOut[matrix_idx]->size, MPI_FLOAT, source, 100, MPI_COMM_WORLD,&requests[i]);
		}


		MPI_Waitall(gpu.MPI_SIZE-1,requests,MPI_STATUSES_IGNORE);
		//hStackN(d_arrA,arrOut[0]->size, out,gpu.MPI_SIZE);

	}
	gpu.tock("chain matrix array");


	gpu.shutdown();



*/
}


void async_test(int argc, char *argv[])
{

	ClusterNet gpu = ClusterNet(argc,argv,1324);
	int rows = 512;
	int cols = 128;


	/*

	MPI_Request r = MPI_REQUEST_NULL;
	MPI_Request s = MPI_REQUEST_NULL;



	Matrix *a = gpu.rand(rows,cols);
	Matrix *b = zeros(rows,cols);

	if(gpu.MYRANK == 0)
	{
		MPI_Irecv(b->data,b->size,MPI_FLOAT,1,0,MPI_COMM_WORLD,&r);
		MPI_Isend(a->data,a->size,MPI_FLOAT,1,0,MPI_COMM_WORLD,&s);
	}
	else
	{
		MPI_Irecv(b->data,b->size,MPI_FLOAT,0,0,MPI_COMM_WORLD,&r);
		MPI_Isend(a->data,a->size,MPI_FLOAT,0,0,MPI_COMM_WORLD,&s);
	}

	MPI_Wait(&s,MPI_STATUS_IGNORE);
	MPI_Wait(&r,MPI_STATUS_IGNORE);


	gpu.tick("MPI");
	for(int i = 0; i < 100; i++)
	{
		if(gpu.MYRANK == 0)
		{
			MPI_Irecv(b->data,b->size,MPI_FLOAT,1,0,MPI_COMM_WORLD,&r);
			MPI_Isend(a->data,a->size,MPI_FLOAT,1,0,MPI_COMM_WORLD,&s);
		}
		else
		{
			MPI_Irecv(b->data,b->size,MPI_FLOAT,0,0,MPI_COMM_WORLD,&r);
			MPI_Isend(a->data,a->size,MPI_FLOAT,0,0,MPI_COMM_WORLD,&s);
		}

		MPI_Wait(&s,MPI_STATUS_IGNORE);
		MPI_Wait(&r,MPI_STATUS_IGNORE);
	}

	gpu.tock("MPI");
	*/





	if(gpu.MYRANK == 0)
	{
		hipSetDevice(0);
		//hipDeviceEnablePeerAccess(1,0);
		hipDeviceDisablePeerAccess(1);
		Matrix *A1 = gpu.rand(rows,cols);
		Matrix *A2 = gpu.rand(rows,cols);
		hipSetDevice(1);
		//hipDeviceEnablePeerAccess(0,0);
		hipDeviceDisablePeerAccess(0);
		Matrix *B1 = gpu.rand(rows,cols);
		Matrix *B2 = gpu.rand(rows,cols);

		hipSetDevice(0);
		hipStream_t s;
		hipStreamCreate(&s);
		hipSetDevice(1);
		hipStream_t s2;
		hipStreamCreate(&s2);
		hipSetDevice(0);

		int access = 0;
		hipDeviceCanAccessPeer(&access,0,1);
		cout << access << endl;
		hipDeviceCanAccessPeer(&access,1,0);
		cout << access << endl;

		hipSetDevice(0);
		gpu.tick("cuda");

		for(int i = 0; i < 100; i++)
		{
			hipMemcpyPeerAsync(B2->data,1,A2->data,0,A2->bytes,s);
			hipSetDevice(1);
			hipMemcpyPeerAsync(A1->data,0,B1->data,1,B1->bytes,s2);

			hipSetDevice(0);
			hipStreamSynchronize(s);
			hipSetDevice(1);
			hipStreamSynchronize(s2);
			hipSetDevice(0);
		}
		gpu.tock("cuda");
	}





	MPI_Barrier(MPI_COMM_WORLD);


	gpu.shutdown_MPI();







}

struct arg_struct
{
		ClusterNet *gpus;
		WikiMaxoutNet *net;
		int device;
};

void *run_net(void * args)
{
	struct arg_struct *_args = (struct arg_struct*)args;
	cout << "device: " << _args->device << endl;
	hipSetDevice(_args->device);
	_args->net->run();

	return 0;
}

void *print_message(void*)
{
    ClusterNet gpu = ClusterNet(124345);
    WikiMaxoutNet net = WikiMaxoutNet(gpu);
    net.run();

    return 0;
}

void bandwidth_test_MPI(int argc, char *argv[])
{
	ClusterNet *gpu = new ClusterNet(argc,argv,1235,true);

	std::vector<MPI_Request*> sends;
	std::vector<MPI_Request*> recvs;
	std::vector<Matrix*> lSync;
	std::vector<Matrix*> lData;

	int packages = 10;
	float time = 0;

	for(int epoch = 1; epoch < 2000; epoch++)
	{
		if(lData.size() > 0)
		{
			for(int i = 0; i < packages; i++)
			{

					hipFree(lSync[i]->data);
					hipFree(lData[i]->data);

			}

			lSync.clear();
			lData.clear();
		}

		for(int i = 0; i < packages; i++)
		{
			lSync.push_back(zeros(128*epoch,128*epoch));
			lData.push_back(gpu->rand(128*epoch,128*epoch));
		}

		for(int j = 0; j < packages; j++)
		{


			MPI_Request *send_request = new MPI_Request;
			MPI_Request *recv_request = new MPI_Request;

			sends.push_back(send_request);
			recvs.push_back(recv_request);

			int target = gpu->MYRANK +1 == gpu->MPI_SIZE ? 0 : gpu->MYRANK+1;
			int source = gpu->MYRANK-1 == -1 ? gpu->MPI_SIZE-1 : gpu->MYRANK-1;

			gpu->tick();
			for (int i = 0; i < gpu->MPI_SIZE -1; i++)
			{
				//MPI_Irecv(lSync[j]->data,lSync[j]->size,MPI_FLOAT,source,999,MPI_COMM_WORLD,recv_request);
				//MPI_Isend(lData[j]->data,lData[j]->size,MPI_FLOAT,target,999,MPI_COMM_WORLD,send_request);
				//MPI_Isend(lData[j]->data,lData[j]->size,MPI_FLOAT,target,j,MPI_COMM_WORLD,send_request);
				if(i == gpu->MYRANK)
				{
					MPI_Send(lData[j]->data,lData[j]->size,MPI_FLOAT,target,j,MPI_COMM_WORLD);
					MPI_Recv(lSync[j]->data,lSync[j]->size,MPI_FLOAT,source,j,MPI_COMM_WORLD, MPI_STATUS_IGNORE);
				}
				else
				{
					MPI_Recv(lSync[j]->data,lSync[j]->size,MPI_FLOAT,source,j,MPI_COMM_WORLD, MPI_STATUS_IGNORE);
					MPI_Send(lData[j]->data,lData[j]->size,MPI_FLOAT,target,j,MPI_COMM_WORLD);
				}


			}
			gpu->tick();


		}


		/*
		gpu->tick();
		for(int i = 0; i < packages; i++)
		{
			MPI_Wait(sends[i],MPI_STATUS_IGNORE);
			MPI_Wait(recvs[i],MPI_STATUS_IGNORE);
		}
		*/
		time = gpu->tock();


		//for(int i = 0; i < packages; i++)
			//assert(sum(lData[i]) == sum(lSync[i]));

		printdim(lData[0]);
		cout << 10*packages*lData[0]->bytes/1024./1024./1024./time << " GB/s" << endl;
	}

	gpu->shutdown_MPI();

}

void bandwidth_test_peer()
{

	ClusterNet *gpu = new ClusterNet(1235);

	std::vector<Matrix*> lSync0;
	std::vector<Matrix*> lData0;
	std::vector<Matrix*> lSync1;
	std::vector<Matrix*> lData1;

	std::vector<hipStream_t> s0s;
	std::vector<hipStream_t> s1s;

	int packages = 1;
	float time = 0;

	hipSetDevice(0);
	hipDeviceEnablePeerAccess(1,0);
	hipSetDevice(1);
	hipDeviceEnablePeerAccess(0,0);
	for(int i = 0; i < packages; i++)
	{
		hipStream_t s0;
		hipStream_t s1;
		hipSetDevice(0);
		hipStreamCreate(&s0);
		hipSetDevice(1);
		hipStreamCreate(&s1);
		s0s.push_back(s0);
		s1s.push_back(s1);
	}
	hipSetDevice(0);
	int access = 0;
	hipDeviceCanAccessPeer(&access,0,1);
	cout << access << endl;
	hipDeviceCanAccessPeer(&access,1,0);
	cout << access << endl;

	for(int epoch = 1; epoch < 100; epoch++)
	{
		if(lSync0.size() > 0)
		{
			for(int i = 0; i < packages; i++)
			{

					hipFree(lSync0[i]->data);
					hipFree(lData0[i]->data);
					hipFree(lSync1[i]->data);
					hipFree(lData1[i]->data);

			}

			lSync0.clear();
			lData0.clear();
			lSync1.clear();
			lData1.clear();
		}

		for(int i = 0; i < packages; i++)
		{
			hipSetDevice(0);
			lSync0.push_back(zeros(128*epoch,128*epoch));
			lData0.push_back(gpu->rand(128*epoch,128*epoch));
			hipSetDevice(1);
			lSync1.push_back(zeros(128*epoch,128*epoch));
			lData1.push_back(gpu->rand(128*epoch,128*epoch));
		}

		hipSetDevice(0);
		gpu->tick();
		for(int j = 0; j < packages; j++)
		{
			hipMemcpyAsync(lSync1[j]->data,lData0[j]->data,lData0[j]->bytes,hipMemcpyDefault, s0s[j]);
			hipSetDevice(1);
			hipMemcpyAsync(lSync0[j]->data,lData1[j]->data,lData1[j]->bytes,hipMemcpyDefault,s1s[j]);
			hipSetDevice(0);
		}





		for(int i = 0; i < packages; i++)
		{
			hipStreamSynchronize(s0s[i]);
			hipStreamSynchronize(s1s[i]);
		}


		time = gpu->tock()/1000.;



		cout << packages*lData0[0]->bytes/1024./1024./1024./time << " GB/s" << endl;
	}

}

void bandwidth_test_kernel()
{

	ClusterNet *gpu = new ClusterNet(1235);

	std::vector<Matrix*> lSync0;
	std::vector<Matrix*> lData0;
	std::vector<Matrix*> lSync1;
	std::vector<Matrix*> lData1;

	std::vector<hipStream_t> s0s;
	std::vector<hipStream_t> s1s;

	int packages = 10;
	float time = 0;

	hipSetDevice(0);
	hipDeviceEnablePeerAccess(1,0);
	hipSetDevice(1);
	hipDeviceEnablePeerAccess(0,0);
	for(int i = 0; i < packages; i++)
	{
		hipStream_t s0;
		hipStream_t s1;
		hipSetDevice(0);
		hipStreamCreate(&s0);
		hipSetDevice(1);
		hipStreamCreate(&s1);
		s0s.push_back(s0);
		s1s.push_back(s1);
	}
	hipSetDevice(0);
	int access = 0;
	hipDeviceCanAccessPeer(&access,0,1);
	cout << access << endl;
	hipDeviceCanAccessPeer(&access,1,0);
	cout << access << endl;

	for(int epoch = 1; epoch < 1000; epoch++)
	{
		if(lSync0.size() > 0)
		{
			for(int i = 0; i < packages; i++)
			{

					hipFree(lSync0[i]->data);
					hipFree(lData0[i]->data);
					hipFree(lSync1[i]->data);
					hipFree(lData1[i]->data);

			}

			lSync0.clear();
			lData0.clear();
			lSync1.clear();
			lData1.clear();
		}

		for(int i = 0; i < packages; i++)
		{
			hipSetDevice(0);
			lSync0.push_back(zeros(128*epoch,128*epoch));
			lData0.push_back(gpu->rand(128*epoch,128*epoch));
			hipSetDevice(1);
			lSync1.push_back(zeros(128*epoch,128*epoch));
			lData1.push_back(gpu->rand(128*epoch,128*epoch));
		}

		hipSetDevice(0);
		gpu->tick();

		for(int j = 0; j < packages; j++)
		{
			add(lSync0[j],lData1[j],lSync0[j]);
			hipSetDevice(1);
			add(lSync1[j],lData0[j],lSync1[j]);
			hipSetDevice(0);
		}

		hipDeviceSynchronize();
		hipSetDevice(1);
		hipDeviceSynchronize();
		hipSetDevice(0);
		time = gpu->tock();

		/*
		for(int i = 0; i < packages; i++)
			assert(sum(lData0[i]) == sum(lSync1[i]));

		for(int i = 0; i < packages; i++)
			assert(sum(lData1[i]) == sum(lSync0[i]));
			*/


		printdim(lSync0[0]);
		cout << 1000*2*packages*lData0[0]->bytes/1024./1024./1024./time << " GB/s" << endl;
	}

}


void bandwidth_test_compression(int argc, char *argv[])
{

	ClusterNet *gpu = new ClusterNet(argc,argv,1235,true);
	MPI_Request *send_request = new MPI_Request;
	MPI_Request *recv_request = new MPI_Request;

	Matrix *w_grad_next = empty(1024,1024);
	Matrix *w_next_sync = empty(1024,1024);

	//warmup
	int target = gpu->MYRANK +1 == gpu->MPI_SIZE ? 0 : gpu->MYRANK+1;
	int source = gpu->MYRANK-1 == -1 ? gpu->MPI_SIZE-1 : gpu->MYRANK-1;

	for (int i = 0; i < gpu->MPI_SIZE - 1; i++)
	{
		MPI_Isend(w_grad_next->data,w_grad_next->size,MPI_FLOAT,target,i,MPI_COMM_WORLD, send_request);
		MPI_Irecv(w_next_sync->data,w_grad_next->size,MPI_FLOAT,source,i,MPI_COMM_WORLD,recv_request);
		target = target +1 == gpu->MPI_SIZE ? 0 : target+1;
		source = source-1 == -1 ? gpu->MPI_SIZE-1 : source-1;
	}

	MPI_Wait(recv_request,MPI_STATUS_IGNORE);

	int times = 100;

	gpu->tick();

	for(int i = 0; i < times; i++)
	{
		target = gpu->MYRANK +1 == gpu->MPI_SIZE ? 0 : gpu->MYRANK+1;
		source = gpu->MYRANK-1 == -1 ? gpu->MPI_SIZE-1 : gpu->MYRANK-1;

		for (int i = 0; i < gpu->MPI_SIZE - 1; i++)
		{
			MPI_Isend(w_grad_next->data,w_grad_next->size,MPI_FLOAT,target,i,MPI_COMM_WORLD, send_request);
			MPI_Recv(w_next_sync->data,w_grad_next->size,MPI_FLOAT,source,i,MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			target = target +1 == gpu->MPI_SIZE ? 0 : target+1;
			source = source-1 == -1 ? gpu->MPI_SIZE-1 : source-1;
		}

		//MPI_Wait(send_request,MPI_STATUS_IGNORE);
	}


	float sec = gpu->tock()*1000.0;
	float GB = 3*times*w_grad_next->bytes/(1024.0*1024.0*1024.0);
	if(gpu->MYRANK == 0)
	{
		cout << "Size in GB: " << GB << endl;
		cout << "GB/s: " << GB/sec << endl;
	}


	gpu->shutdown_MPI();
}

void simple_bandwidth_test(int argc, char *argv[])
{

		ClusterNet *gpu = new ClusterNet(argc,argv,1235,true);
		MPI_Request *send_request = new MPI_Request;
		MPI_Request *recv_request = new MPI_Request;

		int size = 12000;
		for(int i = 8; i < size; i+=8)
		{
			for(int j = 0; j < 3; j++)
			{
				Matrix *w_grad_next;
				if(j==0) w_grad_next = empty(i,i);
				if(j==1) w_grad_next = empty(i/2,i/2);
				if(j==2) w_grad_next = empty(i/4,i/8);
				Matrix *w_next_sync;
				if(j==0) w_next_sync = empty(i,i);
				if(j==1) w_next_sync = empty(i/2,i/2);
				if(j==2) w_next_sync = empty(i/4,i/8);

				if(gpu->MYRANK == 0)
					MPI_Send(w_grad_next->data,w_grad_next->size,MPI_FLOAT,1,999,MPI_COMM_WORLD);
				if(gpu->MYRANK == 1)
					MPI_Recv(w_next_sync->data,w_next_sync->size,MPI_FLOAT,0,999,MPI_COMM_WORLD, MPI_STATUS_IGNORE);

				MPI_Barrier(MPI_COMM_WORLD);
				int times = 100;
				gpu->tick();
				for(int k = 0; k < times; k++)
				{
					if(gpu->MYRANK == 0)
						MPI_Send(w_grad_next->data,w_grad_next->size,MPI_FLOAT,1,999,MPI_COMM_WORLD);
					if(gpu->MYRANK == 1)
						MPI_Recv(w_next_sync->data,w_next_sync->size,MPI_FLOAT,0,999,MPI_COMM_WORLD, MPI_STATUS_IGNORE);

				}

				float quant = 9.5e-08f;
				float dequant = 2.0e-08f;
				float compreess = 1.e-07f;
				float decompress = 5.0e-08f;

				float added_penalty = 0.0f;
				if(j == 1)added_penalty = compreess + decompress;
				if(j == 2)added_penalty = quant + dequant;

				//cout << 100*(added_penalty)*w_grad_next->size << endl;
				float sec = gpu->tock() + (100*(added_penalty)*(i*i));
				float GB = times*w_grad_next->bytes/(1024.0*1024.0*1024.0);
				if(gpu->MYRANK == 0)
				{
					cout << "Size: " << w_grad_next->rows << "x" << w_grad_next->cols << " GB/s: " << GB/(sec/1000) << " " << sec*(j == 2 ? 2.0 : 1.0) << "ms"<< endl;

				}

				hipFree(w_grad_next->data);
				hipFree(w_next_sync->data);
			}
		}


		gpu->shutdown_MPI();

}


void model_parallelism_test(int argc, char *argv[])
{

		ClusterNet *GPU = new ClusterNet(argc,argv,1235,true);
		MPI_Request *send_request = new MPI_Request;
		MPI_Request *recv_request = new MPI_Request;

		std::vector<MPI_Request *> send_requests;
		std::vector<MPI_Request *> recv_requests;

		for(int i = 0; i < GPU->MPI_SIZE-1; i++)
		{
			send_requests.push_back(new MPI_Request);
			recv_requests.push_back(new MPI_Request);
		}

		float max_value = 1.0f;



		for(int round = 128; round <= 8192; round+=128)
		{
			int batch_size = 256;
			int inner = round;
			int outer = round;

			Matrix *A = GPU->rand(batch_size,inner);
			Matrix *B = GPU->distributed_uniformSqrtWeight(inner,outer);
			Matrix *B_normal = GPU->uniformSqrtWeight(inner,outer);
			Matrix *out = zeros(batch_size,outer);
			Matrix *out_stacked = zeros(batch_size,outer);


			int col_split_size = (B->isDistributed == 1 ? B->cols_distributed : B->cols) / GPU->MPI_SIZE;
			int remainder = (B->isDistributed == 1 ? B->cols_distributed : B->cols) - (col_split_size*GPU->MPI_SIZE);

			if(GPU->MYRANK == 0)
			cout << batch_size << "x" << inner << " DOT " << inner << "x" << outer << endl;

			Matrix** arrOut = (Matrix**) malloc(sizeof(Matrix*) * GPU->MPI_SIZE);
			Matrix** arrOut8 = (Matrix**) malloc(sizeof(Matrix*) * GPU->MPI_SIZE);

			for (int i = 0; i < GPU->MPI_SIZE; i++)
			{
				if (i == GPU->MPI_SIZE - 1)
				{
					arrOut[i] = empty(A->rows, col_split_size + remainder);
					arrOut8[i] = empty_char(A->rows, col_split_size + remainder);
				}
				else
				{
					arrOut[i] = empty(A->rows, col_split_size);
					arrOut8[i] = empty_char(A->rows, col_split_size);
				}
			}

			float **h_arrA = (float**) malloc(sizeof(float*) * GPU->MPI_SIZE);
			unsigned char **h_arrA8 = (unsigned char**) malloc(sizeof(unsigned char*) * GPU->MPI_SIZE);
			for (int i = 0; i < GPU->MPI_SIZE; i++)
			{
				h_arrA[i] = arrOut[i]->data;
				h_arrA8[i] = arrOut8[i]->char_data;
			}

			float **d_arrA;
			hipMalloc((void**) &d_arrA, sizeof(float*) * GPU->MPI_SIZE);
			hipMemcpy(d_arrA, h_arrA, sizeof(float*) * GPU->MPI_SIZE,hipMemcpyDefault);

			unsigned char **d_arrA8;
			hipMalloc((unsigned char**) &d_arrA8, sizeof(unsigned char*) * GPU->MPI_SIZE);
			hipMemcpy(d_arrA8, h_arrA8, sizeof(unsigned char*) * GPU->MPI_SIZE,hipMemcpyDefault);

			for(int epoch = 0; epoch < 2; epoch++)
			for(int type = 0; type < 3; type++)
			{
				std::string text = "";
				if(type == 0)
					text = "DOT";
				else if(type == 1)
					text = "DOT32BIT";
				else if(type == 2)
					text = "DOT8BIT";

				if(GPU->MYRANK == 0 && epoch == 1){ GPU->tick(text); }
				for(int i = 0; i < 100; i++)
				{
					if(type == 0)
					{
						GPU->dot(A,B_normal,out);
						continue;
					}
					GPU->dot(A,B,arrOut[GPU->MYRANK]);

					int target = GPU->MYRANK +1 == GPU->MPI_SIZE ? 0 : GPU->MYRANK+1;
					int source = GPU->MYRANK-1 == -1 ? GPU->MPI_SIZE-1 : GPU->MYRANK-1;

					if(type == 2)
					{
						GPU->compression_8bit(arrOut[GPU->MYRANK],max_value,arrOut8[GPU->MYRANK]);
						for (int i = 0; i < GPU->MPI_SIZE - 1; i++)
						{
							MPI_Isend(arrOut8[GPU->MYRANK]->char_data,arrOut8[GPU->MYRANK]->size,MPI_CHAR,target,i,MPI_COMM_WORLD, send_requests[i]);
							MPI_Irecv(arrOut8[source]->char_data,arrOut8[source]->size,MPI_CHAR,source,i,MPI_COMM_WORLD,recv_requests[i]);
							target = target +1 == GPU->MPI_SIZE ? 0 : target+1;
							source = source-1 == -1 ? GPU->MPI_SIZE-1 : source-1;
						}
					}


					if(type == 1)
					{
						for (int i = 0; i < GPU->MPI_SIZE - 1; i++)
						{
							MPI_Isend(arrOut[GPU->MYRANK]->data,arrOut[GPU->MYRANK]->size,MPI_FLOAT,target,i,MPI_COMM_WORLD, send_requests[i]);
							MPI_Irecv(arrOut[source]->data,arrOut[source]->size,MPI_FLOAT,source,i,MPI_COMM_WORLD,recv_requests[i]);
							target = target +1 == GPU->MPI_SIZE ? 0 : target+1;
							source = source-1 == -1 ? GPU->MPI_SIZE-1 : source-1;
						}
					}
					//MPI_Wait(next->send_request,MPI_STATUS_IGNORE);
					for(int i = 0; i < GPU->MPI_SIZE-1; i++)
						MPI_Wait(recv_requests[i],MPI_STATUS_IGNORE);

					if(type == 2)
					{
						for (int i = 0; i < GPU->MPI_SIZE; i++)
						{
							if(i == GPU->MYRANK){continue;}
							GPU->decompression_8bit(arrOut8[i],max_value,arrOut[i]);
						}
					}

					hStackN(d_arrA,	arrOut[0]->size, out_stacked, GPU->MPI_SIZE);


				}
				if(GPU->MYRANK == 0 && epoch == 1){ GPU->tock(text); }


				/*
				MPI_Barrier(MPI_COMM_WORLD);
				if(type == 0)
					printsum(out);
				else if(type == 1)
					printsum(out_stacked);
				else if(type == 2)
					printsum(out_stacked);

				MPI_Barrier(MPI_COMM_WORLD);
				if(type == 0)
					printmat(out,0,4,0,4);
				else if(type == 1)
					printmat(out_stacked,0,4,0,4);
				else if(type == 2)
					printmat(out_stacked,0,4,0,4);

				*/


				if(type == 0)
				{
					abs(out,out);
					max_value = max(out);
				}

			}

			hipFree(A->data);
			hipFree(B->data);
			hipFree(out->data);
			hipFree(out_stacked->data);
			hipFree(d_arrA8);
			hipFree(d_arrA);
			for(int i = 0; i < GPU->MPI_SIZE; i++)
			{
				hipFree(arrOut[i]->data);
				hipFree(arrOut8[i]->char_data);
			}

			size_t total, free;
			hipMemGetInfo(&free, &total);



		}

		GPU->shutdown_MPI();

}

void simple_bandwidth_test_CPU(int argc, char *argv[])
{

		ClusterNet *gpu = new ClusterNet(argc,argv,1235,true);
		MPI_Request *send_request = new MPI_Request;
		MPI_Request *recv_request = new MPI_Request;

		size_t size = 1024*1024*1024;
		float *data = (float*)malloc(sizeof(float)*size);
		float *data_sync = (float*)malloc(sizeof(float)*size);


		int times = 10;
		gpu->tick();
		for(int i = 0; i < times; i++)
		{
			if(gpu->MYRANK == 0)
			{
				MPI_Send(data,size,MPI_FLOAT,1,999,MPI_COMM_WORLD);
				//MPI_Recv(w_next_sync->data,w_next_sync->size,MPI_FLOAT,0,999,MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			}
			if(gpu->MYRANK == 1)
			{
				MPI_Recv(data_sync,size,MPI_FLOAT,0,999,MPI_COMM_WORLD, MPI_STATUS_IGNORE);
				//MPI_Send(w_grad_next->data,w_grad_next->size,MPI_FLOAT,1,999,MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			}
		}


		double sec = gpu->tock()*1000.0;
		double GB = times*size*4.0/(1024.0*1024.0*1024.0);
		if(gpu->MYRANK == 0)
		{
			cout << "Size in GB: " << GB << endl;
			cout << "GB/s: " << GB/sec << endl;
		}


		gpu->shutdown_MPI();

}

void compression_test(int argc, char *argv[])
{

	ClusterNet *gpu = new ClusterNet();
	Matrix *A = scalarMul(gpu->randn(5120,5120),1.0f/10.0f);
	Matrix *out = empty_char(5120,5120);





	gpu->tick();
	for(int i = 0; i < 10000; i++)
		gpu->compression_8bit(A, 0.1f,out);
	gpu->tock();

	gpu->tick();
	for(int i = 0; i < 10000; i++)
		gpu->decompression_8bit(out, 0.1f,A);
	gpu->tock();
}


int main(int argc, char *argv[])
{
	
	//bandwidth_test_peer();

	//bandwidth_test_MPI(argc,argv);

	//bandwidth_test_kernel();

	//compression_test(argc,argv);

	//simple_bandwidth_test(argc,argv);
	//simple_bandwidth_test_CPU(argc,argv);
	//model_parallelism_test(argc,argv);



	//ClusterNet *gpu = new ClusterNet(234);


	/*
	Matrix *rdm = gpu->rand_numbers(10,10);

	printmat(rdm);
	*/


	/*
	ClusterNet *gpu = new ClusterNet(234);
	int out_rows = 128;
	int out_cols = 800;
	int inner = 784;


	Matrix *A = gpu->rand(out_rows,inner);
	Matrix *B = gpu->rand(inner,out_cols);
	Matrix *out1 = zeros(out_rows,out_cols);

	Matrix *charA = empty_char(out_rows,inner);
	Matrix *charB = empty_char(inner,out_cols);
	Matrix *out2 = empty(out_rows,out_cols);
	Matrix *out3 = empty(out_rows,out_cols);

	gpu->tick();
	for(int i = 0; i < 100; i++)
		gpu->dot(A,B,out3);
	gpu->tock();

	float maxA = max(abs(A));
	float maxB = max(abs(B));
	gpu->compression_8bit(A,maxA,charA);
	gpu->compression_8bit(B,maxB,charB);


	//printmat(A);
	//printmat(gpu->decompression_8bit(charA,maxA));
	//printmat(B);
	//printmat(gpu->decompression_8bit(charB,maxB));
	//cout << sum(gpuSqrt(square(sub(B,gpu->decompression_8bit(charB,maxB)))))/(float)B->size << endl;
	//cout << sum(gpuSqrt(square(sub(A,gpu->decompression_8bit(charA,maxA)))))/(float)B->size << endl;
	//gpu->compression_8bit(A,maxA,charA);

	//printmat(out1);
	//printmat(out1,60,65,70,80);
	gpu->tick();
	for(int i = 0; i < 100; i++)
	{
		fill_matrix(out1,0.0f);
		gpu->dot8bit(charA,charB,maxA,maxB,out1);
	}
	gpu->tock();

	gpu->tick();
	for(int i = 0; i < 100; i++)
		gpu->dot8bit_shared(charA,charB,maxA,maxB,out2);
	gpu->tock();
	//printmat(gpu->decompression_8bit(charB,maxB));
	//printmat(out1,60,65,70,80);
	//printmat(out2,60,65,70,80);
	//printmat(out1);
	//printmat(out2);

	//printsum(out1);
	//printsum(out2);
	cout << sum(gpuSqrt(square(sub(out1,out2))))/(float)out1->size << endl;
	cout << sum(gpuSqrt(square(sub(out1,out3))))/(float)out1->size << endl;
	cout << sum(gpuSqrt(square(sub(out2,out3))))/(float)out1->size << endl;


	//cout << "max A " << maxA <<endl;
	//cout << "max B " << maxB <<endl;

	*/



	//ClusterNet *gpu = new ClusterNet(argc,argv,123635,false);
	ClusterNet *gpu = new ClusterNet(argc,argv);
	/*

	Matrix *A = gpu->distributed_uniformSqrtWeight(6,4);
	Matrix *B = gpu->rand(4,6);
	Matrix *A2 = empty(6,2);

	A2->data = A->data;
	printmat(A);

	Matrix *C = gpu->dotMPI(B,A);
	Matrix *C2 = gpu->dot(B,A2);

	printmat(C);
	printmat(C2);

	gpu->shutdown_MPI();
	*/












	//Matrix *X = read_hdf5("/home/tim/data/mnist/X.hdf5");
	//Matrix *y = read_hdf5("/home/tim/data/mnist/y.hdf5");

	Matrix *X = gpu->distribute_file("/home/tim/data/mnist/X.hdf5");
	Matrix *y = gpu->distribute_file("/home/tim/data/mnist/y.hdf5");


	//Matrix *X = gpu->distribute_rows_hdf5_file("/home/tim/data/mnist/X.hdf5");
	//Matrix *y = gpu->distribute_rows_hdf5_file("/home/tim/data/mnist/y.hdf5");
	//Matrix *y = gpu->distribute_rows_hdf5_file("/home/tim/data/mnist/y_15000.hdf5");


	printdim(X);
	printdim(y);
	BatchAllocator b = BatchAllocator();
	//16384
	int batch_size_per_GPU = 128;

	b.init(X,y,(1.0-0.85715),batch_size_per_GPU,128,gpu, Single_GPU);


	Layer *l0 = new Layer(X->cols,batch_size_per_GPU,Input,gpu);
	//l0->PARALLELISM = DataParallelism;
	l0->PARALLELISM = ModelParallelism;
	Layer *l1 = new Layer(1024, Logistic, l0);
	//l1->PARALLELISM = DataParallelism;
	l1->PARALLELISM = ModelParallelism;
	Layer *l2 = new Layer(1024, Logistic, l1);
	//l2->PARALLELISM = DataParallelism;
	l2->PARALLELISM = ModelParallelism;
	Layer *l3 = new Layer(10, Softmax, l2);
	//l3->PARALLELISM = DataParallelism;
	l3->PARALLELISM = ModelParallelism;


	l0->DROPOUT = 0.2f;
	l0->set_hidden_dropout(0.5f);

	cout << gpu->MYRANK << endl;

	float decay = 0.98f;
	gpu->tick("pass");
	b.SKIP_LAST_BATCH = true;
	int epochs = 60;
	for(int epoch = 0; epoch < epochs; epoch++)
	{
		gpu->tick("epoch");
		if(gpu->MYRANK == 0)
			cout << "EPOCH: " << epoch + 1 << endl;
		b.propagate_through_layers(l0,Training,epoch);
		b.propagate_through_layers(l0,Trainerror,epoch);
		b.propagate_through_layers(l0,CVerror,epoch);


		l0->learning_rate_decay(decay);

		if(epoch == 50)
		{
			l0->dropout_decay();
			decay = 0.85f;
		}

		//cout << l1->MAX_GRAD_VALUE << endl;
		gpu->tock("epoch");
	}
	gpu->tock("pass");



	gpu->shutdown_MPI();




	if(gpu->MYRANK == 0)
	{
		int n1 = l3->Train_errors[0].size();
		int n2 = l3->CV_errors[0].size();
		cout << n1 << endl;
		cout << n2 << endl;
		Matrix *train = empty_cpu(epochs,n1);
		Matrix *cv = empty_cpu(epochs,n2);

		for(int i = 0; i < epochs; i++)
		{
			for(int j = 0; j < n1; j++)
				train->data[j + (i*n1)] = l3->Train_errors[i][j];
			for(int j = 0; j < n2; j++)
				cv->data[j + (i*n2)] = l3->CV_errors[i][j];
		}

		write_hdf5("/home/tim/data/mnist/results/32bit/train_error_model.hdf5" ,train);
		write_hdf5("/home/tim/data/mnist/results/32bit/cv_error_model.hdf5",cv);
	}





	/*

	hipSetDevice(0);

	Matrix *X = read_hdf5("/home/tim/data/mnist/X.hdf5");
	Matrix *y = read_hdf5("/home/tim/data/mnist/y.hdf5");



	ClusterNet gpu = ClusterNet(1235);


	BatchAllocator b = BatchAllocator();

	std::vector<int> layers;
	layers.push_back(1200);
	layers.push_back(1200);
	std::vector<float> dropout;
	dropout.push_back(0.2f);
	dropout.push_back(0.5f);
	dropout.push_back(0.5f);
	BatchAllocator allocator = BatchAllocator();
	allocator.init(X,y,(1.0-0.8571429),128,256,gpu, Single_GPU);
	DeepNeuralNetwork net = DeepNeuralNetwork(layers,Classification, gpu, allocator, 10);
	net.EPOCHS = 500;
	net.TRANSITION_EPOCH = 75;
	net.LEARNING_RATE = 0.003;
	net.UPDATE_TYPE = RMSProp;
	net.DROPOUT = dropout;
	//net.MAIN_UNIT = Double_Rectified_Linear;
	net.train();

	*/


	//hipSetDevice(1);
	//ClusterNet *gpus = new ClusterNet(123635);
	//WikiMaxoutNet_PCIe net = WikiMaxoutNet_PCIe(gpus);
	//net.run();


	/*
	hipSetDevice(0);
	struct arg_struct *args0 = (arg_struct*)malloc(sizeof(arg_struct));
	ClusterNet *gpus0 = new ClusterNet(23452345);
	WikiMaxoutNet *net0 = new WikiMaxoutNet(gpus0[0]);
	args0->gpus = gpus0;
	args0->net = net0;
	args0->device = 0;

	net0->run();

	pthread_t t0;
	pthread_create(&t0, NULL, &run_net, args0);

	hipSetDevice(1);
	struct arg_struct *args1 = (arg_struct*)malloc(sizeof(arg_struct));
	ClusterNet *gpus1 = new ClusterNet(23452345);
	WikiMaxoutNet *net1 = new WikiMaxoutNet(gpus1[0]);
	args1->gpus = gpus1;
	args1->net = net1;
	args1->device = 1;

	pthread_t t1;
	//pthread_create(&t1, NULL, &run_net, args1);

	hipSetDevice(2);
	struct arg_struct *args2 = (arg_struct*)malloc(sizeof(arg_struct));
	ClusterNet *gpus2 = new ClusterNet(23452345);
	WikiMaxoutNet *net2 = new WikiMaxoutNet(gpus2[0]);
	args2->gpus = gpus2;
	args2->net = net2;
	args2->device = 2;

	pthread_t t2;
	//pthread_create(&t2, NULL, &run_net, args2);


	cout << "rolfen kek!" << endl;

	void* result0;
	void* result1;
	void* result2;
	pthread_join(t0,&result0);
	//pthread_join(t1,&result1);
	//pthread_join(t2,&result2);
	*/



}







