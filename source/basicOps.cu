#include "hip/hip_runtime.h"
#include <stdio.h>
#include <basicOps.cuh>
#include <clusterKernels.cuh>
#include <assert.h>

Matrix to_gpu(Matrix A){ return to_gpu(A, 0); }
Matrix to_gpu(Matrix A, int is_col_major)
{
  float * gpu_data;
  hipMalloc((void**)&gpu_data,A.bytes);
  hipMemcpy(gpu_data,A.data,A.bytes,hipMemcpyDefault);
  Matrix out = {{A.shape[0],A.shape[1]},A.bytes,A.size,gpu_data};

  if(is_col_major == 0)
	  out = to_col_major(out);

  return out;
}

Matrix to_host(Matrix A){ return to_host(A, 0); }
Matrix to_host(Matrix A, int is_row_major)
{
  Matrix row_major = A;
	 if(is_row_major == 0)
		 row_major = to_row_major(A);
  float *cpu_data;
  cpu_data = (float*)malloc(row_major.bytes);
  hipMemcpy(cpu_data,row_major.data,row_major.bytes,hipMemcpyDefault);
  Matrix out = {{row_major.shape[0],row_major.shape[1]},row_major.bytes,row_major.size,cpu_data};



  return out;
}


static inline void T(Matrix A, Matrix out, int rows, int cols)
{
  // setup execution parameters
  int grid_x = rows / COPY_BLOCK_SIZE;
  if (rows % COPY_BLOCK_SIZE)
    grid_x++;

  int grid_y = cols / COPY_BLOCK_SIZE;
  if (cols % COPY_BLOCK_SIZE)
    grid_y++;

  dim3 grid(grid_x, grid_y, 1);
  dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);
  kTranspose<<< grid, threads >>>(A.data, out.data, rows, cols);

}

Matrix to_col_major(Matrix A)
{
  Matrix out = empty(A.shape[0],A.shape[1]);
  T(A, out, A.shape[1],A.shape[0]);
  //hipFree(A.data);
  return out;
}

Matrix to_row_major(Matrix A)
{
  Matrix out = empty(A.shape[0],A.shape[1]);
  T(A, out, A.shape[0],A.shape[1]);
  //hipFree(A.data);
  return out;
}

Matrix T(Matrix A)
{
  Matrix out = empty(A.shape[1],A.shape[0]);
  T(A, out, A.shape[0],A.shape[1]);

  out.shape[0] = A.shape[1];
  out.shape[1] = A.shape[0];
  return out;
}




Matrix slice_rows(Matrix A, int start, int end)
{
  //align memory in contiguous array

  Matrix out = empty((end - start) + 1, A.shape[1]);
  int block_size = (out.size/1024) + 1;
  slice_rows<<<block_size,1024>>>(A.data, out.data, out.size, A.shape[0], start, end);

  return out;
}

Matrix slice_cols(Matrix A, int start, int end)
{
  Matrix out = empty(A.shape[0], end - start + 1);
  int block_size = (out.size/1024) + 1;
  slice_cols<<<block_size,1024>>>(A.data, out.data, start, A.shape[0], out.size);

  return out;
}

Matrix zeros(int rows, int cols)
{
  return fill_matrix(rows, cols, 0.0f);
}

Matrix ones(int rows, int cols)
{
  return fill_matrix(rows, cols, 1.0f);
}

Matrix empty(int rows, int cols)
{
  float *gpu_data;
  int size = rows*cols;
  size_t bytes = rows*cols*sizeof(float);
  hipMalloc((void**)&gpu_data, bytes);
  
  Matrix A = {{rows, cols}, bytes, size, gpu_data};

  return A;
}


Matrix fill_matrix(int rows, int cols, float fill_value)
{
  if(rows < 1 || cols < 1)
  {
    printf("Error: Dimensions must be greater than zero!\n");
    assert(0);  
  }
 

  float *gpu_data;
  int size = rows*cols;
  size_t bytes = rows*cols*sizeof(float);
  hipMalloc((void**)&gpu_data, bytes);
  
  int block_size = (size/1024) + 1;
  kFill_with<<<block_size,1024>>>(gpu_data, fill_value, size);

  Matrix m = {{rows, cols}, bytes, size, gpu_data};
 
  return m;
}

Matrix add(Matrix A, Matrix B)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  add(A, B, out);
  checkMatrixOperation(A, B, out, 0);

  return out;
}

void add(Matrix A, Matrix B, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kAdd<<<block_size,1024>>>(A.data, B.data, out.data, A.size);
}

Matrix sub(Matrix A, Matrix B)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  sub(A, B, out);
  checkMatrixOperation(A, B, out, 0);

  return out;
}

Matrix vStack(Matrix A, Matrix B)
{

  Matrix out;
  if(A.shape[1] == B.shape[1])
  {
	  out = empty(A.shape[0] + B.shape[0],A.shape[1]);
  }
  else
  {
	  out = empty(1,1);
	  printf("Wrong merge sizes!");
	  assert(0);
  }
  int block_size = (out.size/512) + 1;
  vStack<<<block_size,512>>>(A.data, B.data, out.data, out.size, A.shape[0], A.shape[0] + B.shape[0],A.shape[1]);

  return out;
}

void vStack(Matrix A, Matrix B, Matrix out)
{
  if(A.shape[1] != B.shape[1])
  {
	  printf("Wrong merge sizes!");
	  assert(0);
  }

  int block_size = (out.size/512) + 1;
  vStack<<<block_size,512>>>(A.data, B.data, out.data, out.size, A.shape[0], A.shape[0] + B.shape[0],A.shape[1]);
}

Matrix hStack(Matrix A, Matrix B)
{

  Matrix out;
  if(A.shape[0] == B.shape[0])
  {
	  out = empty(A.shape[0],A.shape[1] + B.shape[1]);
  }
  else
  {
	  out = empty(1,1);
	  printf("Wrong merge sizes!");
	  assert(0);
  }
  int block_size = (out.size/512) + 1;
  hStack<<<block_size,512>>>(A.data, B.data, out.data, out.size, A.size);

  return out;
}

void hStack(Matrix A, Matrix B, Matrix out)
{
  if(A.shape[0] != B.shape[0])
  {
	  printf("Wrong merge sizes!");
	  assert(0);
  }

  int block_size = (out.size/512) + 1;
  hStack<<<block_size,512>>>(A.data, B.data, out.data, out.size, A.size);
}

void sub(Matrix A, Matrix B, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kSub<<<block_size,1024>>>(A.data, B.data, out.data, A.size);
}

Matrix mul(Matrix A, Matrix B)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  mul(A, B, out);
  checkMatrixOperation(A, B, out, 0);

  return out;
}

void mul(Matrix A, Matrix B, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kMul<<<block_size,1024>>>(A.data, B.data, out.data, A.size);
}

Matrix div(Matrix A, Matrix B)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  
  div(A, B, out);
  checkMatrixOperation(A, B, out, 0);

  return out;
}

void div(Matrix A, Matrix B, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kDiv<<<block_size,1024>>>(A.data, B.data, out.data, A.size);
}





Matrix scalarMul(Matrix A, float a)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  scalarMul(A, a, out);

  return out;
}

void scalarMul(Matrix A, float a, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kScalarMul<<<block_size,1024>>>(A.data, a, out.data, A.size);
}

Matrix gpuExp(Matrix A)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  gpuExp(A, out);

  return out;
}

void gpuExp(Matrix A, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kExp<<<block_size,1024>>>(A.data, out.data, A.size);
}

Matrix gpuLog(Matrix A)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  gpuLog(A, out);

  return out;
}

void gpuLog(Matrix A, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kLog<<<block_size,1024>>>(A.data, out.data, A.size);
}

Matrix gpuSqrt(Matrix A)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  gpuSqrt(A, out);

  return out;
}

void gpuSqrt(Matrix A, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kSqrt<<<block_size,1024>>>(A.data, out.data, A.size);
}

Matrix square(Matrix A)
{
  Matrix out = zeros(A.shape[0],A.shape[1]);
  square(A, out);

  return out;
}

void square(Matrix A, Matrix out)
{
  int block_size = (A.size/1024) + 1;
  kSquare<<<block_size,1024>>>(A.data, out.data, A.size);
}

int blnFaultySizes(Matrix A, Matrix B, Matrix C)
{
  if((A.shape[0] == B.shape[0]) &&
     (A.shape[1] == B.shape[1]) &&
     (C.shape[0] == A.shape[0]) &&
     (C.shape[1] == A.shape[1]))
  {
    return 0;
  }
  else
  {
    return 1;
  }
}

int blnFaultyMatrixProductSizes(Matrix A, Matrix B, Matrix C)
{
   if((A.shape[1] == B.shape[0]) &&
      (A.shape[0] == C.shape[0]) &&
      (B.shape[1] == C.shape[1]))
  {
    return 0;
  }
  else
  {
    return 1;
  }
}

void printFaultySizeError(Matrix A, Matrix B, Matrix C)
{
  printf("Error: Faulty matrix sizes:\n");
  if(A.shape[0] != B.shape[0] || A.shape[1] != B.shape[1])
  {
    printf("Matrix A is of size %ix%i while matrix B is of size %ix%i.\n",
           A.shape[0],A.shape[1],B.shape[0],B.shape[1]);
    assert(0);
  }
  else if((A.shape[0] == B.shape[0])  && (A.shape[1] == B.shape[1]) &&          
  	  ((C.shape[0] != A.shape[0]) || (C.shape[1] != A.shape[1])))
  {
    printf("Output matrix is of size %ix%i while the other matrices are of size %ix%i.\n",
           C.shape[0],C.shape[1],B.shape[0],B.shape[1]);
    assert(0);
  }
}
void printFaultyMatrixProductSizeError(Matrix A, Matrix B, Matrix C)
{
    printf("Error: Faulty matrix sizes:\n");  
  if(A.shape[1] != B.shape[0])
  {
    printf("Matrix A is of size %ix%i while matrix B is of size %ix%i.\n",
           A.shape[0],A.shape[1],B.shape[0],B.shape[1]);
  }
  else if((A.shape[1] == B.shape[0])  &&          
  	  ((C.shape[0] != A.shape[0]) || (C.shape[1] != B.shape[1])))
  {
    printf("Output matrix is of size %ix%i while Matrix A and B have sizes %ix%i and %ix%i.\n",
           C.shape[0],C.shape[1],A.shape[0],A.shape[1], B.shape[0],B.shape[1]);
  }
}

int checkMatrixOperation(Matrix A, Matrix B, Matrix C, int blnMatrixProduct)
{
  if(blnMatrixProduct == 0)
  {
    if(blnFaultySizes(A, B, C) == 1)
    {
    	printFaultySizeError(A, B, C);
    	return 1;
    }

  }
  else
  {
    if(blnFaultyMatrixProductSizes(A, B, C) == 1)
    {
      printFaultyMatrixProductSizeError(A, B, C);
      return 1;
    }
  }

  return 0;
}

Matrix softmax(Matrix A)
{
	Matrix out = empty(A.shape[0],A.shape[1]);
	softmax(A, out);
	return out;
}

void softmax(Matrix A, Matrix out)
{
    unsigned int cols = A.shape[1],
                 rows = A.shape[0];



    //if (out.size[0] != h || target->size[1] != w)
    //    return ERROR_INCOMPATIBLE_DIMENSIONS;

    int shared_mem_size = 32 * sizeof(float) ;

    int w1 = floor(sqrt(rows));
    int w2 = rows / w1 + (rows % w1 == 0 ? 0 : 1);
    dim3 gridDim(w1, w2, 1);
    kSoftMax<<<gridDim, 32, shared_mem_size>>>(A.data, out.data, rows, cols);

    hipDeviceSynchronize();

}






