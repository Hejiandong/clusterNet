
#include <hip/hip_runtime.h>
__global__ void kFill_with(float *m, float fill_value, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
       m[i] = fill_value;
  }
}

__global__ void kAdd(float *m1, float *m2, float *m_out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
       m_out[i] = m1[i] + m2[i];
  }
}

__global__ void kMul(float *m1, float *m2, float *m_out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
       m_out[i] = m1[i] * m2[i];
  }
}

__global__ void kSub(float *m1, float *m2, float *m_out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
       m_out[i] = m1[i] - m2[i];
  }
}

__global__ void kDiv(float *m1, float *m2, float *m_out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
       m_out[i] = m1[i] / m2[i];
  }
}
