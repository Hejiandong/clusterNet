#include "hip/hip_runtime.h"
#include <basicOps.cuh>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void kFill_with(float *m, float fill_value, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       m[i] = fill_value;
}

__global__ void kAdd(float *A, float *B, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = A[i] + B[i];
}

__global__ void kMul(float *A, float *B, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = A[i] * B[i];
}

__global__ void kSub(float *A, float *B, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = A[i] - B[i];
}

__global__ void kDiv(float *A, float *B, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = A[i] / B[i];
}

__global__ void kExp(float *A, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = __expf(A[i]);
}

__global__ void kSqrt(float *A, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = sqrt(A[i]);
}

__global__ void kLog(float *A, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = __logf(A[i]);
}

__global__ void kSquare(float *A, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = __powf(A[i], 2);
}

__global__ void kScalarMul(float *A, float scalar, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = scalar*A[i];
}

 
__global__ void kTranspose(float *A, float *out, int width, int height) 
{
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < width) && (yIndex < height)) 
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = A[index_in];
    }

    __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < height) && (yIndex < width)) 
    {
        unsigned int index_out = yIndex * height + xIndex;
        out[index_out] = block[threadIdx.x][threadIdx.y];
    }
}










