#include "hip/hip_runtime.h"
#include <basicOps.cuh>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

__global__ void kFill_with(float *m, float fill_value, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       m[i] = fill_value;
}

__global__ void kMerge(float *A, float *B, float *out, int size_a, int size_b)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  const int size = size_a + size_b;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
    if(i >= size_a)
    {
       out[i] = B[i - size_a];
    }
    else
    {
       out[i] = A[i];
    }
  }
}

__global__ void kAdd(float *A, float *B, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = A[i] + B[i];
}

__global__ void kMul(float *A, float *B, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = A[i] * B[i];
}

__global__ void kSub(float *A, float *B, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = A[i] - B[i];
}

__global__ void kDiv(float *A, float *B, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = A[i] / B[i];
}

__global__ void kExp(float *A, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = __expf(A[i]);
}

__global__ void kSqrt(float *A, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = sqrt(A[i]);
}

__global__ void kLog(float *A, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = __logf(A[i]);
}

__global__ void kSquare(float *A, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = __powf(A[i], 2);
}

__global__ void kScalarMul(float *A, float scalar, float *out, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
       out[i] = scalar*A[i];
}

 
__global__ void kTranspose(float *A, float *out, int width, int height) 
{
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < width) && (yIndex < height)) 
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = A[index_in];
    }

    __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < height) && (yIndex < width)) 
    {
        unsigned int index_out = yIndex * height + xIndex;
        out[index_out] = block[threadIdx.x][threadIdx.y];
    }
}


__global__ void slice_rows(float *A, int start, int end, int cols, float *out)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  int offset = start*cols;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x + offset;
  int slice_end = (end*cols);

  for (unsigned int i = idx;i < slice_end; i += numThreads)
       out[i-offset] = A[i];
}

__global__ void slice_cols(float *A, int start, int end, int rows, int cols, float *out)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  const int width = end - start;  
  int current_row = 0;
  int size = cols*rows;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
     if(((i % cols) >= start) && ((i % cols) < end))
     {
       current_row = i/cols;       
       out[(current_row*width) + (i % cols) - start] = A[i];
     }   
  }
}


