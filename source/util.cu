#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <list>
#include <vector>
#include <util.cuh>
#include <basicOps.cuh>
#include <hdf5.h>
#include <iomanip>
#include <ios>
#include <assert.h>

using std::string;
using std::vector;
using std::cout;
using std::endl;

Matrix *read_csv (const char* filename)
{
  std::ifstream  dStream(filename);
  int columns = 0;
  int rows = 0;
  vector<float> X;

    string line;
    while(std::getline(dStream,line))
    {
        std::stringstream  lineStream(line);
        string        cell;
        while(std::getline(lineStream,cell,','))
        {
			X.push_back(::atof(cell.c_str()));

			if(rows == 0)
				columns++;
        }
	rows++;
    }

  float *data;
  size_t bytes = columns*rows*sizeof(float);
  hipHostAlloc(&data, bytes, hipHostMallocPortable);
  memcpy(data,&X[0], columns*rows*sizeof(float));

  Matrix *out = (Matrix*)malloc(sizeof(Matrix));
  out->rows = rows;
  out->cols = columns;
  out->bytes = bytes;
  out->size = columns*rows;
  out->data = data;
  out->isDistributed = 0;
  out->cols_distributed = 0;
  out->isSparse = 0;

  return out;
}

void write_csv(const char* filename, Matrix *X, const char* header, Matrix *ids)
{
	std::ofstream myfile;
	myfile.open(filename,std::ios::trunc);
	myfile << header << "\r\n";
	for(int row = 0; row< X->rows; row++)
	  {
		  for(int col = 0; col < X->cols; col++)
		  {
			  if(col > 0)
				  myfile << ",";
			  else
				  myfile << (int)ids->data[row] << ",";

			  myfile << std::fixed << X->data[(row*X->cols)+col];
		  }
		  myfile << "\r\n";
	  }
	myfile.close();
}

void write_csv(const char* filename, Matrix *X)
{
	std::ofstream myfile;
	myfile.open(filename,std::ios::trunc);
	for(int row = 0; row< X->rows; row++)
	  {
		  for(int col = 0; col < X->cols; col++)
		  {
			  if(col > 0)
				  myfile << ",";

			  myfile << std::fixed << X->data[(row*X->cols)+col];
		  }
		  myfile << "\r\n";
	  }
	myfile.close();
}

Matrix *read_hdf5(const char *filepath){ return read_hdf5(filepath,"/Default"); }
Matrix *read_hdf5(const char *filepath, const char *tag)
{
	   hid_t       file_id, dataset_id;

	   file_id = H5Fopen(filepath, H5F_ACC_RDWR, H5P_DEFAULT);
	   dataset_id = H5Dopen2(file_id, tag, H5P_DEFAULT);

	   hid_t dspace = H5Dget_space(dataset_id);
	   hsize_t dims[2];
	   H5Sget_simple_extent_dims(dspace, dims, NULL);
	   size_t bytes = sizeof(float)*dims[0]*dims[1];

	   float *data;
	   hipHostAlloc(&data, bytes, hipHostMallocPortable);

	   H5Dread(dataset_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, data);
	   H5Dclose(dataset_id);
	   H5Fclose(file_id);

	   Matrix *out = (Matrix*)malloc(sizeof(Matrix));
	   out->rows = (int)dims[0];
	   out->cols= (int)dims[1];
	   out->bytes = bytes;
	   out->data = data;
	   out->size = (int)(dims[0]*dims[1]);
	   out->isDistributed = 0;
	   out->cols_distributed = 0;
	   out->isSparse = 0;

	   return out;
}

Matrix *read_sparse_hdf5(const char *filepath)
{
	hid_t       file_id, dataset_id_idx, dataset_id_ptr, dataset_id_data, dataset_id_shape, dspace;
	hsize_t dims[2];
	size_t bytes;
	file_id = H5Fopen(filepath, H5F_ACC_RDWR, H5P_DEFAULT);
	Matrix *out = (Matrix*)malloc(sizeof(Matrix));

	dataset_id_idx = H5Dopen2(file_id, "/indices", H5P_DEFAULT);
	dspace = H5Dget_space(dataset_id_idx);
	H5Sget_simple_extent_dims(dspace, dims, NULL);
	bytes = sizeof(int)*dims[0];
	int *idx;
	hipHostAlloc(&idx, bytes, hipHostMallocPortable);
	H5Dread(dataset_id_idx, H5T_NATIVE_INT, H5S_ALL, H5S_ALL, H5P_DEFAULT, idx);
	H5Dclose(dataset_id_idx);

	out->idx_bytes = sizeof(int)*dims[0];
	out->idx_cols = idx;


	dataset_id_ptr = H5Dopen2(file_id, "/indptr", H5P_DEFAULT);
	dspace = H5Dget_space(dataset_id_ptr);
	H5Sget_simple_extent_dims(dspace, dims, NULL);
	bytes = sizeof(int)*dims[0];
	int *ptr;
	hipHostAlloc(&ptr, bytes, hipHostMallocPortable);
	H5Dread(dataset_id_ptr, H5T_NATIVE_INT, H5S_ALL, H5S_ALL, H5P_DEFAULT, ptr);
	H5Dclose(dataset_id_ptr);

	out->ptr_bytes = sizeof(int)*dims[0];
	out->ptr_rows = ptr;


	dataset_id_data = H5Dopen2(file_id, "/data", H5P_DEFAULT);
	dspace = H5Dget_space(dataset_id_data);
	H5Sget_simple_extent_dims(dspace, dims, NULL);
	bytes = sizeof(float)*dims[0];
	float *data;
	hipHostAlloc(&data, bytes, hipHostMallocPortable);
	H5Dread(dataset_id_data, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, data);
	H5Dclose(dataset_id_data);

	out->bytes = sizeof(float)*dims[0];
	out->size = (int)dims[0];

	dataset_id_shape = H5Dopen2(file_id, "/shape", H5P_DEFAULT);
	dspace = H5Dget_space(dataset_id_shape);
	H5Sget_simple_extent_dims(dspace, dims, NULL);
	bytes = sizeof(long)*dims[0];
	long shape[2];
	H5Dread(dataset_id_shape, H5T_NATIVE_LONG, H5S_ALL, H5S_ALL, H5P_DEFAULT, shape);
	H5Dclose(dataset_id_shape);

	H5Fclose(file_id);


	out->rows = (int)shape[0];
	out->cols= (int)shape[1];
	out->data = data;
	out->isDistributed = 0;
	out->isSparse = 1;




	return out;
}

void write_hdf5(const char * filepath, Matrix *A)
{
	   hid_t       file_id, dataset_id, dataspace_id;
	   hsize_t     dims[2];

	   file_id = H5Fcreate(filepath, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
	   dims[0] = A->rows;
	   dims[1] = A->cols;
	   dataspace_id = H5Screate_simple(2, dims, NULL);
	   dataset_id = H5Dcreate2(file_id, "/Default", H5T_NATIVE_FLOAT, dataspace_id, H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT);

	   H5Dwrite(dataset_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, A->data);
	   H5Dclose(dataset_id);
	   H5Fclose(file_id);
}

hipEvent_t* tick()
{
    hipEvent_t* startstop;
    startstop = (hipEvent_t*)malloc(2*sizeof(hipEvent_t));
    hipEventCreate(&startstop[0]);
    hipEventCreate(&startstop[1]);
    hipEventRecord(startstop[0], 0);

    return startstop;
}

void tock(hipEvent_t* startstop){ tock(startstop, "Time for the kernel(s): "); }
void tock(hipEvent_t* startstop, std::string text)
{
	float time;
	hipEventRecord(startstop[1], 0);
	hipEventSynchronize(startstop[1]);
	hipEventElapsedTime(&time, startstop[0], startstop[1]);
	printf((text + ": %f ms.\n").c_str(), time);
}
void tock(std::string text, float tocks)
{
	printf((text + ": %f ms.\n").c_str(), tocks);
}
float tock(hipEvent_t* startstop, float tocks)
{
	float time;
	hipEventRecord(startstop[1], 0);
	hipEventSynchronize(startstop[1]);
	hipEventElapsedTime(&time, startstop[0], startstop[1]);

	return time+tocks;
}



int test_eq(float f1, float f2, char* message)
{
  if(f1 == f2){ return 1;}
  else{ printf("%s: %f != %f\n", message, f1, f2); }
  return 0;
}

int test_eq(float f1, float f2, int idx1, int idx2, char* message)
{
  if(f1 == f2){ return 1;}
  else{ printf("%s: %f != %f for index %i and %i.\n", message, f1, f2, idx1, idx2); }
  return 0;
}

int test_eq(int i1, int i2, char* message)
{
  if(i1 == i2){ return 1;}
  else{ printf("%s: %i != %i\n", message, i1, i2); }
  return 0;
}

int test_eq(int i1, int i2, int idx1, int idx2, char* message)
{
  if(i1 == i2){ return 1;}
  else{ printf("%s: %i != %i for index %i and %i.\n", message, i1, i2, idx1, idx2); }
  return 0;
}

int test_matrix(Matrix *A, int rows, int cols)
{
  if((A->rows == rows) &&
     (A->cols == cols) &&
     (A->size == cols*rows) &&
     (A->bytes == cols*rows*sizeof(float)))
      {return 1;}
  else
  {
    test_eq(A->rows,rows,"Matrix rows");
    test_eq(A->cols,cols,"Matrix cols");
    test_eq(A->size,cols*rows,"Matrix size");
    test_eq((int)(A->bytes),(int)(cols*rows*sizeof(float)),"Matrix bytes");
  }

  return 0;
}

void print_matrix(Matrix *A, int end_rows, int end_cols)
{
	if(A->isSparse != 1)
	{
		for(int row = 0; row< end_rows; row++)
		  {
			  printf("[");
			  for(int col =0; col < end_cols; col++)
			  {
				  printf("%f ",A->data[(row*A->cols)+col]);
			  }
			  printf("]\n");
		  }
		  printf("\n");
	}
	else
	{
		printf("[");
		for(int i = end_rows; i < end_cols; i++)
			printf("%f ",A->data[i]);

		printf("]\n");
	}
}

void print_matrix(Matrix *A, int start_row, int end_row, int start_col, int end_col)
{
	assert(A->isSparse == 0);

	for(int row = start_row; row< end_row; row++)
	{
		printf("[");
		for(int col =start_col; col < end_col; col++)
		{
			printf("%f ",A->data[(row*A->cols)+col]);
		}
		printf("]\n");
	}
	printf("\n");

}

void printmat(Matrix *A)
{
  Matrix * m = to_host(A);
  if(A->isSparse == 0)
	  print_matrix(m,A->rows,A->cols);
  else
	  print_matrix(m,0,A->size);
  free(m->data);
  free(m);

}

void printdim(Matrix *A)
{
	cout << A->rows << "x" << A->cols << endl;
}

void printsum(Matrix *A)
{
	cout << sum(A) << endl;
}

void printhostmat(Matrix *A)
{
  if(A->isSparse == 0)
	  print_matrix(A,A->rows,A->cols);
  else
	  print_matrix(A,0,A->size);
}

void printmat(Matrix *A, int end_rows, int end_cols)
{
  Matrix * m = to_host(A);
  print_matrix(m, end_rows, end_cols);
  free(m->data);
  free(m);

}

void printmat(Matrix *A, int start_row, int end_row, int start_col, int end_col)
{
  Matrix * m = to_host(A);
  print_matrix(m, start_row, end_row, start_col, end_col);
  free(m->data);
  free(m);

}

bool replace(std::string& str, const std::string& from, const std::string& to)
{
    size_t start_pos = str.find(from);
    if(start_pos == std::string::npos)
        return false;
    str.replace(start_pos, from.length(), to);
    return true;
}

void slice_sparse_to_dense(Matrix *X, Matrix *out, int start, int length)
{
	int idx_from = 0;
	int idx_to = 0;
	int idx = 0;

	for(int i = 0; i < out->size; i++)
		out->data[i] = 0.0f;

	for(int row = 0; row < length; row++)
	{
		idx_from = X->ptr_rows[start + row];
		idx_to = X->ptr_rows[start + row + 1];

		for(int i = idx_from; i < idx_to; i++)
		{
			idx = X->idx_cols[i];
			out->data[(row*out->cols) + idx] = X->data[i];
		}
	}



}

float determine_max_sparsity(Matrix *X, int batch_size)
{

	float max_sparsity = 0.0;

	Matrix *dense_batch = empty_cpu(batch_size,X->cols);
	int batches = (X->rows / batch_size);
	float batch_elements = batch_size*X->cols;

	float nonzero_count = 0.0f;
	for(int i = 0; i < batches; i++)
	{
		nonzero_count = (X->ptr_rows[(i+1)*batch_size] - X->ptr_rows[i*batch_size]);

		if(max_sparsity < (nonzero_count / batch_elements))
			max_sparsity = (nonzero_count / batch_elements);

		nonzero_count = 0.0f;
	}

	return max_sparsity;

}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}




