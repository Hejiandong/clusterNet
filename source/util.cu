#include <stdio.h>
#include <clusterNet.cuh>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <list>
#include <vector>
#include <util.cuh>
#include <basicOps.cuh>

using std::string;
using std::vector;

Matrix read_csv (char* filename)
{
  std::ifstream  dStream(filename);
  int dimX = 0;
  int dimY = 0;
  vector<float> X;

    string line;
    while(std::getline(dStream,line))
    {
        std::stringstream  lineStream(line);
        string        cell;
        while(std::getline(lineStream,cell,','))
        {
	    X.push_back(::atof(cell.c_str()));
	    
	if(dimY == 0)
	    dimX++;
        }
	dimY++;
    }

  
  float *data;  
  data = (float*)malloc(dimX*dimY*sizeof(float));
  memcpy(data,&X[0], dimX*dimY*sizeof(float));
  Matrix m = {{dimX,dimY},dimX*dimY*sizeof(float),dimX*dimY,data};  

  return m;
}


hipEvent_t* tick()
{
    hipEvent_t* startstop;
    startstop = (hipEvent_t*)malloc(2*sizeof(hipEvent_t));
    hipEventCreate(&startstop[0]);
    hipEventCreate(&startstop[1]);
    hipEventRecord(startstop[0], 0);

    return startstop;
}

void tock(hipEvent_t* startstop)
{
    float time;
    hipEventRecord(startstop[1], 0);
    hipEventSynchronize(startstop[1]);   
    hipEventElapsedTime(&time, startstop[0], startstop[1]);
    printf ("Time for the kernel: %f ms\n", time);
}

