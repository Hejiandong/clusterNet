#include <basicOps.cuh>
#include <cudaLibraryOps.cuh>
#include <clusterKernels.cuh>
#include <hipblas.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <util.cuh>



Matrix dot(Matrix A, Matrix B)
{
  Matrix out = zeros(A.shape[0],B.shape[1]);
  dot(A, B, out);
  
  checkMatrixOperation(A, B, out, 1);

  return out;
}

void dot(Matrix A, Matrix B, Matrix out)
{	
  hipblasStatus_t status;
	
  const float alpha = 1.0f;
  const float beta = 0.0f;

  hipblasHandle_t h;
  hipblasCreate(&h);      
    
  status = hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                A.shape[0], B.shape[1], A.shape[1],
                &alpha, A.data, A.shape[0],
                B.data, B.shape[0],
                &beta, out.data, out.shape[0]);   
                
  if(status != HIPBLAS_STATUS_SUCCESS)
    printf("CUBLAS ERROR!");
}

hiprandGenerator_t random_init(){ return random_init(time(0)); }
hiprandGenerator_t random_init(int seed)
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);    
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandSetGeneratorOffset(gen, 100);

    return gen;
}

Matrix rand(hiprandGenerator_t gen, int rows, int cols)
{ 
    Matrix out = empty(rows, cols);
    rand(gen, rows, cols, out);

    return out;
}
void rand(hiprandGenerator_t gen, int rows, int cols, Matrix out){ hiprandGenerateUniform(gen, out.data, rows*cols); }

Matrix randn(hiprandGenerator_t gen, int rows, int cols){ return randn(gen, rows, cols, 0, 1); }
Matrix randn(hiprandGenerator_t gen, int rows, int cols, float mean, float std)
{
    Matrix out = empty(rows,cols);  
    randn(gen, rows, cols, mean, std, out);
    
    return out;
}
void randn(hiprandGenerator_t gen, int rows, int cols, float mean, float std, Matrix out){ hiprandGenerateNormal(gen, out.data, rows*cols, 0.0f, 1.0f); }

