#include "hip/hip_runtime.h"
#include <basicOps.cuh>
#include <clusterNet.h>
#include <batchAllocator.h>
#include <assert.h>
#include <stdio.h>
#include <util.cuh>
#include <batchAllocator.h>


int run_clusterNet_test(ClusterNet gpus)
{
	ClusterNet gpu = ClusterNet();
	ClusterNet ticktock_test = ClusterNet();
	ticktock_test.tick("ClusterNet test ran in");

	//dot test
	//      0 2    3             17 0
	// m1 = 0 0.83 59.1387  m2 =  3 4
	//                            0 0

	//row major data
	float m1_data[6] = {0,2,3,0,0.83,59.1387};
	float m2_data[6] = {17,0,3,4,0,0};
	size_t m1_bytes = 2*3*sizeof(float);
	Matrix *m1_cpu = (Matrix*)malloc(sizeof(Matrix));
	m1_cpu->rows = 2;
	m1_cpu->cols = 3;
	m1_cpu->bytes = m1_bytes;
	m1_cpu->size = 6;
	m1_cpu->data = m1_data;
	Matrix *m2_cpu = (Matrix*)malloc(sizeof(Matrix));
	m2_cpu->rows = 3;
	m2_cpu->cols = 2;
	m2_cpu->bytes = m1_bytes;
	m2_cpu->size = 6;
	m2_cpu->data = m2_data;
	Matrix *m1 = to_gpu(m1_cpu);
	Matrix *m2 = to_gpu(m2_cpu);

	//dense to sparse and to_host for sparse matrix test
	Matrix *s1 = gpus.dense_to_sparse(m1);
	Matrix *m_host = to_host(s1);

	ASSERT(s1->rows == 2, "empty sparse rows");
	ASSERT(s1->cols == 3, "empty sparse cols");
	ASSERT(s1->size == 4, "empty sparse size");
	ASSERT(s1->isSparse == 1, "empty sparse");
	ASSERT(s1->idx_bytes == sizeof(float)*4, "empty sparse bytes");
	ASSERT(s1->bytes == sizeof(float)*4, "empty sparse bytes");
	ASSERT(s1->ptr_bytes == sizeof(float)*(s1->rows + 1), "empty sparse bytes");
	assert(test_eq(m_host->data[0], 2.0f,"sparse data."));
	assert(test_eq(m_host->data[1], 3.0f,"sparse data."));
	assert(test_eq(m_host->data[2], 0.83f,"sparse data."));
	assert(test_eq(m_host->data[3], 59.1387f,"sparse data."));


	m_host = to_host(gpus.sparse_to_dense(s1));
	assert(test_eq(m_host->data[0], 0.0f,"sparse to dense data."));
	assert(test_eq(m_host->data[1], 2.0f,"sparse to dense data."));
	assert(test_eq(m_host->data[2], 3.0f,"sparse to dense data."));
	assert(test_eq(m_host->data[3], 0.0f,"sparse to dense data."));
	assert(test_eq(m_host->data[4], 0.83f,"sparse to dense data."));
	assert(test_eq(m_host->data[5], 59.1387f,"sparse to dense data."));

	m_host = to_host(m1);

	//dot test
	Matrix *m3 = gpu.dot(m1,m2);
	Matrix *out = zeros(2,2);
	m_host = to_host(m3);

	assert(test_eq(m_host->data[0], 6.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 8.0f,"Dot data."));
	assert(test_eq(m_host->data[2], 2.49f,"Dot data."));
	assert(test_eq(m_host->data[3], 3.32f,"Dot data."));
	assert(test_matrix(m_host,2,2));

	gpu.dot(m1,m2,out);
	m_host = to_host(out);
	assert(test_eq(m_host->data[0], 6.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 8.0f,"Dot data."));
	assert(test_eq(m_host->data[2], 2.49f,"Dot data."));
	assert(test_eq(m_host->data[3], 3.32f,"Dot data."));
	assert(test_matrix(m_host,2,2));

	//dot sparse test
	m3 = gpu.dot_sparse(s1,m2);
	m_host = to_host(m3);

	assert(test_eq(m_host->data[0], 6.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 8.0f,"Dot data."));
	assert(test_eq(m_host->data[2], 2.49f,"Dot data."));
	assert(test_eq(m_host->data[3], 3.32f,"Dot data."));
	assert(test_matrix(m_host,2,2));

	out = empty(2,2);
	gpu.dot_sparse(s1,m2, out);
	m_host = to_host(out);

	assert(test_eq(m_host->data[0], 6.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 8.0f,"Dot data."));
	assert(test_eq(m_host->data[2], 2.49f,"Dot data."));
	assert(test_eq(m_host->data[3], 3.32f,"Dot data."));
	assert(test_matrix(m_host,2,2));

	//Tdot test

	out = zeros(3,3);
	gpu.Tdot(m1,m1,out);
	m_host = to_host(out);
	assert(test_eq(m_host->data[0], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[2], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[3], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[4], 4.6889f,"Dot data."));
	assert(test_eq(m_host->data[5], 55.085117f,"Dot data."));
	assert(test_eq(m_host->data[6], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[7], 55.085117f,"Dot data."));
	assert(test_eq(m_host->data[8], 3506.385742f,"Dot data."));
	assert(test_matrix(m_host,3,3));

	out = zeros(2,2);
	gpu.Tdot(m2,m2,out);
	m_host = to_host(out);
	assert(test_eq(m_host->data[0], 298.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 12.0f,"Dot data."));
	assert(test_eq(m_host->data[2], 12.0f,"Dot data."));
	assert(test_eq(m_host->data[3], 16.0f,"Dot data."));
	assert(test_matrix(m_host,2,2));

	//Tdot sparse test

	out = zeros(3,3);
	assert(out->rows == s1->cols && out->cols == m1->cols);
	gpu.Tdot_sparse(s1,m1,out);
	m_host = to_host(out);
	assert(test_eq(m_host->data[0], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[2], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[3], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[4], 4.6889f,"Dot data."));
	assert(test_eq(m_host->data[5], 55.085117f,"Dot data."));
	assert(test_eq(m_host->data[6], 0.0f,"Dot data."));
	assert(test_eq(m_host->data[7], 55.085117f,"Dot data."));
	assert(test_eq(m_host->data[8], 3506.385742f,"Dot data."));
	assert(test_matrix(m_host,3,3));

	out = zeros(2,2);
	Matrix *s5 = gpus.dense_to_sparse(m2);
	assert(out->rows == s5->cols && out->cols == m2->cols);
	gpu.Tdot_sparse(s5,m2,out);
	m_host = to_host(out);
	assert(test_eq(m_host->data[0], 298.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 12.0f,"Dot data."));
	assert(test_eq(m_host->data[2], 12.0f,"Dot data."));
	assert(test_eq(m_host->data[3], 16.0f,"Dot data."));
	assert(test_matrix(m_host,2,2));
	//dot T test
	gpu.dotT(m1,m1,out);
	m_host = to_host(out);
	assert(test_eq(m_host->data[0], 13.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 179.0761f,"Dot data."));
	assert(test_eq(m_host->data[2], 179.0761f,"Dot data."));
	assert(test_eq(m_host->data[3], 3498.074463f,"Dot data."));
	assert(test_matrix(m_host,2,2));

	//dot T sparse test
	out = zeros(2,2);
	gpu.dotT_sparse(s1,m1,out);
	m_host = to_host(out);
	assert(test_eq(m_host->data[0], 13.0f,"Dot data."));
	assert(test_eq(m_host->data[1], 179.0761f,"Dot data."));
	assert(test_eq(m_host->data[2], 179.0761f,"Dot data."));
	assert(test_eq(m_host->data[3], 3498.074463f,"Dot data."));
	assert(test_matrix(m_host,2,2));

	//test uniform random
	Matrix *r1 = gpu.rand(100,100);
	m_host = to_host(r1);
	int upper = 0;
	int lower = 0;
	int zeros = 0;
	for(int i = 0; i < r1->size; i++)
	{
	assert(m_host->data[i] >= 0.0f);
	assert(m_host->data[i] <= 1.0f);
	if(m_host->data[i] > 0.5f)
	   upper++;
	else
	   lower++;

	if(m_host->data[i] == 0)
	   zeros++;
	}
	//there should be more than 47% which is > 0.5
	assert(upper > (r1->size)*0.47f);
	assert(lower > (r1->size)*0.47f);
	assert(m_host->rows==100);
	assert(m_host->cols==100);
	assert(m_host->size==100*100);
	assert(m_host->bytes==r1->size*sizeof(float));

	//test same seeds
	ClusterNet gpu2 = ClusterNet(1234);
	gpu = ClusterNet(1234);
	r1 = gpu.rand(10,10);
	Matrix *r2 = gpu2.rand(10,10);
	Matrix *h1 = to_host(r1);
	Matrix *h2 = to_host(r2);
	for(int i = 0; i < 100; i++)
	{
	assert(h1->data[i] == h2->data[i]);
	}
	//test different seeds
	gpu2 = ClusterNet(1235);
	gpu = ClusterNet(1234);
	r1 = gpu.rand(10,10);
	r2 = gpu2.rand(10,10);
	h1 = to_host(r1);
	h2 = to_host(r2);
	for(int i = 0; i < 100; i++)
	{
	assert(h1->data[i] != h2->data[i]);
	}

	//test normal random
	r1 = gpu.randn(100,479);
	m_host = to_host(r1);
	upper = 0;
	lower = 0;
	int middle = 0;
	zeros = 0;
	for(int i = 0; i < r1->size; i++)
	{
	if(m_host->data[i] > 1.96f)
	   upper++;

	if(m_host->data[i] < -1.96f)
	   lower++;

	if(m_host->data[i] == 0)
	   zeros++;

	if((m_host->data[i] < 1) && (m_host->data[i] > -1))
	   middle++;
	}
	//printmat(r1);
	//a z-score of greater than 1.96 should only occur with 2.5% probability
	assert(upper < r1->size*0.04);
	assert(lower < r1->size*0.04);
	//the 68% of the data should be within one standard deviation
	assert((middle > r1->size*0.65) && (middle < r1->size*0.70));
	//if there are more than 1% zeros then there is something fishy
	assert(zeros < r1->size*0.01);
	assert(test_matrix(m_host,100,479));

	//dotMPI test

	m1 = scalarMul(ones(200,400),0.3);
	m2 = scalarAdd(ones(400,800),0.1748345);

	m3 = gpus.dot(m1,m2);
	Matrix *m4 = gpus.dotMPI(m1,m2);
	m3 = to_host(m3);
	m4 = to_host(m4);
	if(gpus.MYRANK == 0)
	{

	  for (int i = 0; i < m3->size; ++i)
	  {
		  assert(test_eq(m3->data[i],m4->data[i],i,i,"dotMPI Test"));
	  }

	  assert(test_matrix(m3,200,800));
	  assert(test_matrix(m4,200,800));
	}

	int count = 0;
	//distributed weights test
	m_host = to_host(gpus.distributed_uniformSqrtWeight(10000,1000));
	if(gpus.MYRANK < gpus.MPI_SIZE-1)
		assert(test_matrix(m_host,10000,1000/gpus.MPI_SIZE));
	else
		assert(test_matrix(m_host,10000,1000-((1000/gpus.MPI_SIZE)*(gpus.MPI_SIZE-1))));
	count = 0;
	for(int i = 0; i < m_host->size; i++)
	{
	  ASSERT((m_host->data[i] > -4.0f*sqrt(6.0f/(10000.0+1000.0))) && (m_host->data[i] < 4.0f*sqrt(6.0f/(10000.0+1000.0))),"Distributed RdmSqrtWeight test");
	  if(m_host->data[i] == 0)
		  count++;
	}
	ASSERT(count < 10,"Distributed RdmSqrtWeight test");

	count = 0;
	m_host = to_host(gpus.distributed_uniformSqrtWeight(100,10));
	if(gpus.MYRANK < gpus.MPI_SIZE-1)
		assert(test_matrix(m_host,100,10/gpus.MPI_SIZE));
	else
		assert(test_matrix(m_host,100,10-((10/gpus.MPI_SIZE)*(gpus.MPI_SIZE-1))));
	count = 0;
	for(int i = 0; i < m_host->size; i++)
	{
	  ASSERT((m_host->data[i] > -4.0f*sqrt(6.0f/(100.0+10.0))) && (m_host->data[i] < 4.0f*sqrt(6.0f/(100.0+10.0))),"Distributed RdmSqrtWeight test");
	  if(m_host->data[i] == 0)
		  count++;
	}
	ASSERT(count < 10,"Distributed RdmSqrtWeight test");

	m1 = gpus.distributed_uniformSqrtWeight(7833,83);
	test_eq(m1->rows,7833,"distributed rdmsqrt split size test");
	if(gpus.MYRANK < gpus.MPI_SIZE-1)
		test_eq(m1->cols,83/gpus.MPI_SIZE,"distributed rdmsqrt split size test");
	else
		test_eq(m1->cols,83-((83/gpus.MPI_SIZE)*(gpus.MPI_SIZE-1)),"distributed rdmsqrt split size test");


	//distributed zeros test
	m_host = to_host(gpus.distributed_zeros(10000,1000));
	if(gpus.MYRANK < gpus.MPI_SIZE-1)
		assert(test_matrix(m_host,10000,1000/gpus.MPI_SIZE));
	else
		assert(test_matrix(m_host,10000,1000-((1000/gpus.MPI_SIZE)*(gpus.MPI_SIZE-1))));
	for(int i = 0; i < m_host->size; i++)
	{
	  ASSERT(m_host->data[i] == 0.0f,"Distributed zeros test");
	}

	m_host = to_host(gpus.distributed_zeros(100,10));
	if(gpus.MYRANK < gpus.MPI_SIZE-1)
		assert(test_matrix(m_host,100,10/gpus.MPI_SIZE));
	else
		assert(test_matrix(m_host,100,10-((10/gpus.MPI_SIZE)*(gpus.MPI_SIZE-1))));
	for(int i = 0; i < m_host->size; i++)
	{
	  ASSERT(m_host->data[i] == 0.0f,"Distributed zeros test");
	}

	//dotMPI test for distributed weights
	m1 = gpus.distributed_zeros(8783,317);
	scalarAdd(m1,1.0,m1);
	m2 = ones(111,8783);
	m3 = ones(17,317);
	m4 = ones(8783,17);
	m_host = to_host(m1);
	for(int i = 0; i < m_host->size; i++)
	{
		assert(test_eq(m_host->data[i],1.0f,"dotMPI test"));
	}
	for(int epoch = 0; epoch < 5; epoch++)
	{
		//indirect dotMPIs
		m_host = to_host(gpus.dot(m2,m1));
		assert(test_matrix(m_host,111,317));
		for(int i = 0; i < m_host->size; i++)
		{
			assert(test_eq(m_host->data[i],8783.0f,"dotMPI test"));
		}
		m_host = to_host(gpus.dotTMPI(m3,m1));
		assert(test_matrix(m_host,17,8783));
		for(int i = 0; i < m_host->size; i++)
		{
			assert(test_eq(m_host->data[i],317.0f,"dotMPI test"));
		}
		gpus.dot(m4,m3,m1);
		m_host = to_host(m1);
		for(int i = 0; i < m_host->size; i++)
		{
			assert(test_eq(m_host->data[i],17.0f,"dotMPI test"));
		}
		//direct dotMPIs
		m1 = gpus.distributed_zeros(8783,317);
		scalarAdd(m1,1.0,m1);
		m_host = to_host(gpus.dotMPI(m2,m1));
		assert(test_matrix(m_host,111,317));
		for(int i = 0; i < m_host->size; i++)
		{
			assert(test_eq(m_host->data[i],8783.0f,"dotMPI test"));
		}
		m_host = to_host(gpus.dotTMPI(m3,m1));
		assert(test_matrix(m_host,17,8783));
		for(int i = 0; i < m_host->size; i++)
		{
			assert(test_eq(m_host->data[i],317.0f,"dotMPI test"));
		}
		gpus.dotMPI(m4,m3,m1);
		m_host = to_host(m1);
		for(int i = 0; i < m_host->size; i++)
		{
			assert(test_eq(m_host->data[i],17.0f,"dotMPI test"));
		}
		m1 = gpus.distributed_zeros(8783,317);
		scalarAdd(m1,1.0,m1);
	}

	m1 = gpus.distributed_zeros(8783,317);
	scalarAdd(m1,1.0,m1);
	m2 = gpus.dropout(gpus.rand(111,8783),0.5);
	m3 = gpus.dropout(gpus.rand(17,317),0.5);
	m4 = gpus.dropout(gpus.rand(8783,17),0.5);
	Matrix *m5 = gpus.dropout(gpus.rand(111,317),0.5);


	Matrix *s2 = gpus.dense_to_sparse(m2);
	gpus.dropout(s2,0.5);
	m2 = gpus.sparse_to_dense(s2);
	Matrix *s3 = gpus.dense_to_sparse(m3);
	Matrix *s4 = gpus.dense_to_sparse(m4);
	Matrix *sHost;
	s5 = gpus.dense_to_sparse(m5);
	for(int epoch = 0; epoch < 5; epoch++)
	{
		//indirect dotMPIs
		m_host = to_host(gpus.dot(m2,m1));
		sHost = to_host(gpus.dot(s2,m1));
		assert(test_matrix(m_host,111,317));
		for(int i = 0; i < m_host->size; i++)
		{
			ASSERT(m_host->data[i] + 0.1 > sHost->data[i] &&
				   m_host->data[i] - 0.1 < sHost->data[i],"dotMPI test");
		}
		/*
		 * NOT SUPPORTED YET
		 *
		m_host = to_host(gpus.dotTMPI(m3,m1));
		sHost = to_host(gpus.dotT(s3,m1));
		for(int i = 0; i < m_host->size; i++)
		{
			printf("%f vs %f\n",m_host->data[i],sHost->data[i]);
			ASSERT(m_host->data[i] + 0.1 > sHost->data[i] &&
				   m_host->data[i] - 0.1 < sHost->data[i],"dotMPI test");
		}
		*/

		gpus.dot(m4,m3,m1);
		m_host = to_host(m1);
		gpus.dot(s4,m3,m1);
		sHost = to_host(m1);
		for(int i = 0; i < m_host->size; i++)
		{
			ASSERT(m_host->data[i] + 0.1 > sHost->data[i] &&
				   m_host->data[i] - 0.1 < sHost->data[i],"dotMPI test");
		}

		gpus.Tdot(m2,m5,m1);
		m_host = to_host(m1);
		gpus.Tdot(s2,m5,m1);
		sHost = to_host(m1);
		for(int i = 0; i < m_host->size; i++)
		{
			if(0 == (m_host->data[i] + 0.1 > sHost->data[i] &&
				   m_host->data[i] - 0.1 < sHost->data[i]))
				printf("%f vs %f\n",m_host->data[i],sHost->data[i]);

			ASSERT(m_host->data[i] + 0.1 > sHost->data[i] &&
				   m_host->data[i] - 0.1 < sHost->data[i],"dotMPI test");
		}

		m1 = gpus.distributed_zeros(8783,317);
		scalarAdd(m1,1.0,m1);
		m4 = gpus.dropout(gpus.rand(8783,17),0.5);
		s4 = gpus.dense_to_sparse(m4);
	}



	//dropout test
	m1 = gpu.rand(1000,1000);
	m_host = to_host(gpu.dropout(m1,0.5));
	assert(test_matrix(m_host,1000,1000));
	count = 0;
	for(int i = 0; i < m1->size; i++)
	{
	   if(m_host->data[i] == 0.0f)
	   count++;
	}
	ASSERT((count >= 499000) && (count < 501000),"dropout test");
	m1 = gpu.rand(1000,1000);
	m_host = to_host(gpu.dropout(m1,0.2));
	count = 0;
	for(int i = 0; i < m1->size; i++)
	{
	   if(m_host->data[i] == 0.0f)
	   count++;
	}
	ASSERT((count >= 199000) && (count < 201000),"dropout test");
	m1 = gpu.rand(1000,1000);
	m_host = to_host(gpu.dropout(m1,0.73));
	count = 0;
	for(int i = 0; i < m1->size; i++)
	{
	   if(m_host->data[i] == 0.0f)
	   count++;
	}
	ASSERT((count >= 729000) && (count < 731000),"dropout test");



	//rdmsqrtweight test
	m1 = gpu.uniformSqrtWeight(784,777);
	m_host = to_host(m1);
	assert(test_matrix(m_host,784,777));
	count = 0;
	for(int i = 0; i < m1->size; i++)
	{
	  ASSERT((m_host->data[i] > -4.0f*sqrt(6.0f/(784.0+777.0))) && (m_host->data[i] < 4.0f*sqrt(6.0f/(784.0+777.0))),"RdmSqrtWeight test");
	  if(m_host->data[i] == 0)
		  count++;
	}

	ASSERT(count < 10,"RdmSqrtWeight test");

	//rand_int test
	m1 = gpu.rand_int(784,777,2,10);
	m_host = to_host(m1);
	assert(test_matrix(m_host,784,777));
	count = 0;
	for(int i = 0; i < m1->size; i++)
	{
	  ASSERT((m_host->data[i] >= 2) && (m_host->data[i] <= 10),"rand_int test");
	  if(m_host->data[i] == 0)
		  count++;
	}

	m1 = gpu.rand_int(100,100,782965,78254609);
	m_host = to_host(m1);
	assert(test_matrix(m_host,100,100));
	count = 0;
	for(int i = 0; i < m1->size; i++)
	{
	  ASSERT((m_host->data[i] >= 782965) && (m_host->data[i] <= 78254609),"rand_int test");
	  if(m_host->data[i] == 0)
		  count++;
	}
	ASSERT(count == 0,"rand_int test");

	m_host = to_host(gpu.rand_int(1000,1000,0,9));
	int counts[10] = {0,0,0,0,0,
					  0,0,0,0,0};
	assert(test_matrix(m_host,1000,1000));
	for(int i = 0; i < m_host->size; i++)
	{
		counts[(int)m_host->data[i]]++;
	}
	for(int i = 0; i < 10; i++)
	{
		//expectation is 100000 each
		ASSERT((counts[i] > 95000) && (counts[i] < 105000), "rand_int test");
	}

	//rdmsparseweight test
	m1 = gpu.sparseInitWeight(784,812);
	m_host = to_host(m1,1);
	assert(test_matrix(m_host,784,812));
	count = 0;
	for(int i = 0; i < m_host->size; i++)
	{
		if(m_host->data[i] != 0.0f)
		{
			count++;
		}
	}
	//average should be bigger than 14
	ASSERT(count/812.0f > 14.0f,"sparse weight test");
	m1 = gpu.sparseInitWeight(532,2000,73);
	m_host = to_host(m1);
	assert(test_matrix(m_host,532,2000));
	count = 0;
	for(int i = 0; i < m_host->size; i++)
	{
		if(m_host->data[i] != 0.0f)
		{
			count++;
		}
	}
	//average should be bigger than 65 (there is a high chance of re-rolling the same number)
	ASSERT(count/2000.0f > 65.0f,"sparse weight test");

	//Tdot_sparse test for large matricies
	m1 = gpus.dropout(gpus.rand(128,9000),0.9);
	s1 = gpus.dense_to_sparse(m1);
	m2 = gpus.rand(128,100);
	m3 = empty(9000,100);

	gpus.Tdot(m1,m2,m3);
	m_host = to_host(m3);
	gpus.Tdot(s1,m2,m3);
	sHost = to_host(m3);
	for(int i = 0; i < sHost->size; i++)
		ASSERT(m_host->data[i] + 0.1 > sHost->data[i] &&
			   m_host->data[i] - 0.1 < sHost->data[i],"large matrix sparse Tdot test.");


	//sparse to dense equal test
	m1 = gpus.dropout(gpus.rand(128,9000),0.9);
	s1 = gpus.dense_to_sparse(m1);
	assert(test_eq((int)s1->ptr_bytes,(int)129*sizeof(int),"dense to sparse dimension test"));
	s3 = to_host(gpus.dense_to_sparse(m1));
	assert(test_eq((int)s3->ptr_bytes,(int)129*sizeof(int),"dense to sparse dimension test"));
	s4 = to_host(gpus.dense_to_sparse(gpus.sparse_to_dense(s1)));
	assert(test_eq((int)s4->ptr_bytes,(int)129*sizeof(int),"dense to sparse dimension test"));
	m2 = to_host(gpus.sparse_to_dense(s1));
	m1 = to_host(m1);

	for(int i = 0; i < m1->size; i++)
		assert(test_eq(m1->data[i],m2->data[i],"dense to sparse and back equality."));

	for(int i = 0; i < s3->size; i++)
	{
		assert(test_eq(s3->data[i],s4->data[i],"dense to sparse and back equality."));
		assert(test_eq(s3->idx_cols[i],s4->idx_cols[i],"dense to sparse and back equality."));
	}

	for(int i = 0; i < s3->rows + 1; i++)
		assert(test_eq(s3->ptr_rows[i],s4->ptr_rows[i],"dense to sparse and back equality."));


	//construct vocab test
	int vocab_vector_size = 37;
	int batch_size = 137;
	int window_size = 17;
	int middleIdx = window_size/2;
	int vocab_size = 333;
	Matrix *vocab_idx = gpu.rand_int(batch_size,window_size,0,vocab_size-1);
	Matrix *vocab = gpu.uniformSqrtWeight(vocab_vector_size,vocab_size);
	Matrix *batch_X = gpu.rand(batch_size,vocab_vector_size*window_size);
	Matrix *batch_Y = gpu.rand(batch_size,vocab_vector_size*window_size);
	gpu.construct_vocab_matrix(vocab_idx,batch_X,batch_Y,vocab);

	m1 = to_host(vocab_idx);
	m2 = to_host(vocab);
	m3 = to_host(batch_X);
	m4 = to_host(batch_Y);

	int idx = 0;
	for(int row = 0; row < vocab_idx->rows; row++)
		for(int col = 0; col < vocab_idx->cols; col++)
		{
			idx = (int)m1->data[col + (row*m1->cols)];
			for(int i = 0; i < vocab_vector_size; i++)
			{
				assert(test_eq(m3->data[(col*vocab_vector_size) + (row*batch_X->cols) + i],m2->data[idx + (vocab->cols*i)],"construct vocab matrix test"));
				if(col != middleIdx)
					assert(test_eq(m4->data[(col*vocab_vector_size) + (row*batch_X->cols) + i],m2->data[idx + (vocab->cols*i)],"construct vocab matrix test"));
				else
					ASSERT(m4->data[(col*vocab_vector_size) + (row*batch_X->cols) + i] != m3->data[(col*vocab_vector_size) + (row*batch_X->cols) + i] &&
							(m4->data[(col*vocab_vector_size) + (row*batch_X->cols) + i] < 50.0 &&
							 m4->data[(col*vocab_vector_size) + (row*batch_X->cols) + i] > -50.0),"construct vocab matrix test");//if the number is >50 or < -50 something is wrong

			}
		}


	//This should just pass without error
	ticktock_test.tock("ClusterNet test ran in");

	return 0;
}


