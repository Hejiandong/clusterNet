#include "hip/hip_runtime.h"
#include <stdio.h>
#include <mpi.h>
#include <assert.h>
//#include <basicOps.cuh>
#include <math.h>
#include <util.cuh>
#include <clusterNet.h>

int run_basicOps_test()
{

  ClusterNet gpu = ClusterNet();

  Matrix *m1 = ones(5,6);
  Matrix *m2 = ones(5,6);
  Matrix *m3 = zeros(5,6);
  Matrix *out = zeros(5,6);
  
  //to_col_major test
  //      0 2    3             
  // m1 = 0 0.83 59.1387  
  //                           
  float m1_data[6] = {0,2,3,0,0.83,59.1387};
  size_t m1_bytes = 2*3*sizeof(float);
  Matrix *m1_cpu = (Matrix*)malloc(sizeof(Matrix));
  m1_cpu->rows = 2;
  m1_cpu->cols = 3;
  m1_cpu->bytes = m1_bytes;
  m1_cpu->size = 6;
  m1_cpu->data = m1_data;

  m1 = to_gpu(m1_cpu,1);
  //to_col_major test
  m1 = to_col_major(m1);
  float *test;
  test = (float*)malloc(m1->bytes);
  hipMemcpy(test,m1->data,m1->bytes,hipMemcpyDefault);

  assert(test_eq(test[0], 0.0f,"To col major data."));
  assert(test_eq(test[1], 0.0f,"To col major data."));
  assert(test_eq(test[2], 2.0f,"To col major data."));
  assert(test_eq(test[3], 0.83f,"To col major data."));
  assert(test_eq(test[4], 3.0f,"To col major data."));
  assert(test_eq(test[5], 59.1387f,"To col major data."));



   m1 = to_row_major(m1);
   hipMemcpy(test,m1->data,m1->bytes,hipMemcpyDefault);

   assert(test_eq(test[0], 0.0f,"To row major data."));
   assert(test_eq(test[1], 2.0f,"To row major data."));
   assert(test_eq(test[2], 3.0f,"To row major data."));
   assert(test_eq(test[3], 0.0f,"To row major data."));
   assert(test_eq(test[4], 0.83f,"To row major data."));
   assert(test_eq(test[5], 59.1387f,"To row major data."));

   assert(test_eq(getNonZeroElements(m1),4 ,"Get non-zero elements."));


  //test to_host
  //data is converted to column major and then back to row major
  Matrix *m_host = to_host(to_gpu(m1_cpu));
  assert(m_host->rows==m1->rows);
  assert(m_host->cols==m1->cols);
  assert(m_host->size==m1->size);
  assert(m_host->bytes==m1->bytes);
  for(int i = 0; i< 5; i++)
  {
    assert(m_host->data[i]==m1_cpu->data[i]);
  }


  //test fill_with
  m1 = ones(5,6);
  m_host = to_host(m1);
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==1.0f);
  }

  //test add
  m3 = add(m1,m2);
  m_host = to_host(m3);
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==2.0f);
  } 

  //test to_gpu
  m_host =  to_host(add(to_gpu(m_host),to_gpu(m_host)));
  for(int i = 0; i< 30; i++)
  {
    assert(test_eq(m_host->data[i],4.0f,"To gpu data"));
  } 

  //test mul
  m3 = mul(m3,m3);
  m_host = to_host(m3);
  for(int i = 0; i< 30; i++)
  {
    assert(test_eq(m_host->data[i],4.0f,"Multiplication data"));
  } 

  //test sub
  m3 = sub(m3,m1);
  m_host = to_host(m3);
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==3.0f);
  } 

  //test div
  m2 = add(m1,m2); //2
  m3 = div(m3,m2);
  m_host = to_host(m3);
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==1.5f);
  } 

  //test add with given output Matrix *
  add(m3,m2,out);
  m_host = to_host(out);
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==3.5f);
  }

  //test sub with given output Matrix *
  sub(m3,m2,out);
  m_host = to_host(out);
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==-0.5f);
  }

  //test mul with given output Matrix *
  mul(m3,m2,out);
  m_host = to_host(out);
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==3.0f);
  }

  //test div with given output Matrix *
  div(m3,m2,out);
  m_host = to_host(out);
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==0.75f);
  }
  
  //test exp
  m_host = to_host(gpuExp(zeros(5,6)));
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==1.0f);
  }

  //test scalar mul
  m_host = to_host(scalarMul(ones(5,6),1.83));
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==1.83f);
  }

  //test sqrt
  m_host = to_host(gpuSqrt(scalarMul(ones(5,6),4)));
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==2.0f);
  }

  //test log
  m_host = to_host(gpuLog(scalarMul(ones(5,6),2.0)));
  for(int i = 0; i< 30; i++)
  {   
    assert(m_host->data[i]==log(2.0f));
  }

  //test square
  m_host = to_host(square(scalarMul(ones(5,6),2)));
  for(int i = 0; i< 30; i++)
  {
    assert(m_host->data[i]==4.0f);
  }

  //test blnFaultySizes
  assert(blnFaultySizes(ones(1,3),ones(2,3),ones(2,3))==1);
  assert(blnFaultySizes(ones(1,3),ones(1,3),ones(2,3))==1);
  assert(blnFaultySizes(ones(1,3),ones(1,3),ones(1,3))==0);
  assert(blnFaultySizes(ones(3,3),ones(3,3),ones(3,3))==0);
  //test blnFaultyMatrixSizes
  assert(blnFaultyMatrixProductSizes(ones(1,3),ones(1,3),ones(3,3))==1);
  assert(blnFaultyMatrixProductSizes(ones(3,1),ones(1,3),ones(2,2))==1);
  assert(blnFaultyMatrixProductSizes(ones(3,1),ones(1,3),ones(3,3))==0);

  //transpose test
  //column major order
  //      0 2    3
  // m1 = 0 0.83 59.1387
  //
  //test to_gpu with is_col_major = 1
  m_host = to_host(T(to_gpu(m1_cpu)));
  assert(test_eq(m_host->data[0],0.0f,"Transpose data."));
  assert(m_host->data[1]==0.0f);
  assert(m_host->data[2]==2.0f);
  assert(m_host->data[3]==0.83f);
  assert(m_host->data[4]==3.0f);
  assert(m_host->data[5]==59.1387f);
  assert(test_matrix(m_host,3,2));

  //to host and to gpu test
  //      0 2    3
  // m1 = 0 0.83 59.1387
  //
  //to gpu and to host should cancel each other out
  m_host = to_host(to_gpu(m1_cpu));
  assert(m_host->data[0]==0.0f);
  assert(m_host->data[1]==2.0f);
  assert(m_host->data[2]==3.0f);
  assert(m_host->data[3]==0.0f);
  assert(m_host->data[4]==0.83f);
  assert(m_host->data[5]==59.1387f);
  assert(test_matrix(m_host,2,3));

  //to_gpu for col major data test
  //col major data
  float m2_data[6] = {0,0,2,0.83,3,59.1387};
  size_t m2_bytes = 2*3*sizeof(float);
  Matrix *m2_cpu = (Matrix*)malloc(sizeof(Matrix));
  m2_cpu->rows = 2;
  m2_cpu->cols = 3;
  m2_cpu->bytes = m2_bytes;
  m2_cpu->size = 6;
  m2_cpu->data = m2_data;
  m_host = to_host(to_gpu(m2_cpu,1));
  //should be in row major now
  assert(m_host->data[0]==0.0f);
  assert(m_host->data[1]==2.0f);
  assert(m_host->data[2]==3.0f);
  assert(m_host->data[3]==0.0f);
  assert(m_host->data[4]==0.83f);
  assert(m_host->data[5]==59.1387f);
  assert(test_matrix(m_host,2,3));


  //slice rows
  m1 = gpu.rand(10,10);
  m2 = to_host(slice_rows(m1, 2,5));
  m1 = to_host(m1);
  assert(test_matrix(m2,4,10));
  int idx = 0;
  for(int i = 20; i < 60; i++)
  {        
    assert(test_eq(m1->data[i], m2->data[idx], idx, i , "Row slice data"));
    idx++;
  }  

  //slice cols
  m1 = gpu.rand(10,10);
  m2 = to_host(slice_cols(m1, 2,5));
  m1 = to_host(m1);
  idx = 0;
  assert(test_matrix(m2,10,4));


  for(int i = 2; i < 100;i++)
  {
    if(((i % 10) < 6) &&
       ((i % 10) > 1))
    {  
      assert(test_eq(m1->data[i], m2->data[idx], idx, i , "Col slice data"));
      idx++;
    }
  }

  //softmax test
  m1 = softmax(ones(2056,10));
  m_host = to_host(m1);
  assert(test_matrix(m_host,2056,10));
  for(int i = 0; i < m_host->size; i++)
  {
	  assert(test_eq(m_host->data[i],0.1,"Softmax equal test"));
  }

  m1 = softmax(gpu.rand(2222,17));
  m_host = to_host(m1);
  assert(test_matrix(m_host,2222,17));
  float sum_value = 0;
  for(int i = 0; i < m_host->size; i++)
  {
	  sum_value += m_host->data[i];
	  if((i > 0) &&  (((i+1) % 17) == 0))
	  {
		  ASSERT((sum_value > 0.99) && (sum_value < 1.01), "Softmax row sum equal one");
		  sum_value = 0.0f;
	  }
  }


  m1 = zeros(10,10);
  m2 = ones(10,1);
  //sub matrix vector test: A - v
  m_host= to_host(subMatrixVector(m1,m2));
  assert(test_matrix(m_host,10,10));
  for(int i = 0; i < m_host->size; i++)
  {
	  assert(test_eq(m_host->data[i],-1.0f, "Matrix - vector, equal data test"));
  }
  m3 = gpu.rand(13,17);
  Matrix *m4 = gpu.rand(1,17);
  m_host = to_host(addMatrixVector(m3,m4));
  m3 = to_host(m3);
  m4 = to_host(m4);
  assert(test_matrix(m_host,13,17));
  for(int row = 0; row < m_host->rows; row++)
  {
	  for(int col = 0; col < m_host->cols; col++)
		  assert(test_eq(m_host->data[(row*m_host->cols) + col], m3->data[(row*m_host->cols) + col] + m4->data[col], "Matrix + vector, equal data test"));
  }

  //      0 2    3
  // m1 = 0 0.83 59.1387
  //
  //argmax test
  //col_value = A[(i*cols) + idx];
  m1 = argmax(to_gpu(m1_cpu));
  m_host = to_host(m1);
  assert(test_matrix(m_host,2,1));
  assert(test_eq(m_host->data[0],2.0f, "Argmax test"));
  assert(test_eq(m_host->data[1],2.0f, "Argmax test"));
  m1 = gpu.rand(2056,10);
  m_host = to_host(argmax(m1));
  int counts[10] = {0,0,0,0,0,
		  	  	  	0,0,0,0,0};
  assert(test_matrix(m_host,2056,1));
  for(int i = 0; i < m_host->size; i++)
  {
	  counts[(int)m_host->data[i]]++;
  }
  for(int i = 0; i < 10; i++)
  {
	  //expectation is 205.6 each;
	  ASSERT((counts[i] > 140) && (counts[i] < 280), "Argmax value test");
  }

  //create t matrix test
  m1 = scalarMul(ones(10,1),4);
  m1 = create_t_matrix(m1,7);
  m_host = to_host(m1);
  assert(test_matrix(m_host,10,7));
  for(int i = 0; i < m_host->size; i++)
  {
	  if((i % m1->cols) == 4)
	  {
		  assert(test_eq(m_host->data[i],1.0f, "Create t matrix data"));
	  }
	  else
	  {
		  assert(test_eq(m_host->data[i],0.0f, "Create t matrix data"));
	  }
  }

  //equal test
  gpu = ClusterNet(12345);
  ClusterNet gpu2 = ClusterNet(12345);
  m2 = gpu.rand(10,7);
  m1 = gpu2.rand(10,7);
  m_host = to_host(equal(m1,m2));
  assert(test_matrix(m_host,10,7));
  for(int i = 0; i < m_host->size; i++)
  {
	  assert(test_eq(m_host->data[i],1.0f, "Matrix matrix Equal data test"));
  }
  m1 = gpu2.rand(10,7);
  m_host = to_host(equal(m1,m2));
  assert(test_matrix(m_host,10,7));
  for(int i = 0; i < m_host->size; i++)
  {
	  assert(test_eq(m_host->data[i],0.0f, "Matrix matrix Equal data test"));
  }


  //test sum
  m1 = ones(10,1);
  m2 = ones(1,10);

  ASSERT(sum(m1) == 10.0f, "Vector sum test");
  ASSERT(sum(m2)  == 10.0f, "Vector sum test");
  m1 = ones(10,10);
  ASSERT(sum(m1)  == 100.0f, "Vector sum test");
  ASSERT(sum(scalarMul(m2,1.73)) > 17.29f, "Vector sum test");
  ASSERT(sum(scalarMul(m2,1.73)) < 17.31f, "Vector sum test");

  //logistic test
  m1 = zeros(2,2);
  m1 = to_host(logistic(m1));
  assert(test_matrix(m1,2,2));
  for(int i = 0; i < m1->size; i++)
  {
	  ASSERT(m1->data[i] == 0.5f,"Logistic data test.");
  }
  m1 = gpu.randn(100,100);
  m1 = to_host(logistic(m1));
  assert(test_matrix(m1,100,100));
  for(int i = 0; i < m1->size; i++)
  {
	  ASSERT((m1->data[i] > 0.0f) && (m1->data[i] < 1.0f),"Logistic data test.");
  }

  //logistic grad test
  m1 = ones(2,2);
  m1 = to_host(logisticGrad(m1));
  assert(test_matrix(m1,2,2));
  for(int i = 0; i < m1->size; i++)
  {
	  ASSERT(m1->data[i] == 0.0f,"Logistic data test.");
  }
  m1 = gpu.randn(100,100);
  m_host = to_host(m1);
  m1 = to_host(logisticGrad(m1));
  assert(test_matrix(m1,100,100));
  for(int i = 0; i < m1->size; i++)
  {
	  ASSERT(m_host->data[i]*(1-m_host->data[i]) == m1->data[i],"Logistic data test.");
  }

  //arange test
  m1 = arange(10,7);
  m_host = to_host(m1);
  assert(test_matrix(m_host,10,7));
  for(int i = 0; i < m1->size; i++)
  {
	  assert(test_eq(m_host->data[i],(float)i, "Arange data test."));
  }

  m1 = arange(101,10,7);
  m_host = to_host(m1);
  assert(test_matrix(m_host,10,7));
  for(int i = 0; i < m1->size; i++)
  {
	  assert(test_eq(m_host->data[i],(float)(i + 101), "Arange data test."));
  }

  //cutoff to probability test
  m_host = to_host(doubleRectifiedLinear(gpu.randn(123,357,0,10)));
  assert(test_matrix(m_host,123,357));
  for(int i = 0; i < m_host->size; i++)
	  ASSERT((m_host->data[i] <=1.0f) && (m_host->data[i] >=0.0f),"cutoff to probability test.");


  m1 = empty_sparse(17,83,0.01783,0.0);
  int elements = ceil(17*83*0.01783) + 1;
  ASSERT(m1->rows == 17, "empty sparse rows");
  ASSERT(m1->cols == 83, "empty sparse cols");
  ASSERT(m1->size == elements, "empty sparse size");
  ASSERT(m1->isSparse == 1, "empty sparse");
  ASSERT(m1->idx_bytes == sizeof(float)*elements, "empty sparse bytes");
  ASSERT(m1->bytes == sizeof(float)*elements, "empty sparse bytes");
  ASSERT(m1->ptr_bytes == sizeof(float)*(m1->rows + 1), "empty sparse bytes");

  m1 = empty_sparse(17,83,500);
  elements = 500;
  ASSERT(m1->rows == 17, "empty sparse rows");
  ASSERT(m1->cols == 83, "empty sparse cols");
  ASSERT(m1->size == elements, "empty sparse size");
  ASSERT(m1->isSparse == 1, "empty sparse");
  ASSERT(m1->idx_bytes == sizeof(float)*elements, "empty sparse bytes");
  ASSERT(m1->bytes == sizeof(float)*elements, "empty sparse bytes");
  ASSERT(m1->ptr_bytes == sizeof(float)*(m1->rows + 1), "empty sparse bytes");

  m1 = empty_pinned_sparse(171,837,0.01783,0.001110);
  elements = ceil(171*837*(0.01783+0.001110)) + 1;
  ASSERT(m1->rows == 171, "empty sparse rows");
  ASSERT(m1->cols == 837, "empty sparse cols");
  ASSERT(m1->size == elements, "empty sparse size");
  ASSERT(m1->isSparse == 1, "empty sparse");
  ASSERT(m1->idx_bytes == sizeof(float)*elements, "empty sparse bytes");
  ASSERT(m1->bytes == sizeof(float)*elements, "empty sparse bytes");
  ASSERT(m1->ptr_bytes == sizeof(float)*(m1->rows + 1), "empty sparse bytes");

  for(int i = 0; i < m1->size; i++)
  {
	  ASSERT(m1->data[i] == 0.0f,"empty sparse data");
	  ASSERT(m1->idx_cols[i] == 0.0f,"empty sparse data");
  }




  return 0;
}



