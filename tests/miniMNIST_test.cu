#include <stdio.h>
#include <util.cuh>
#include <basicOps.cuh>
#include <assert.h>
#include <clusterNet.h>
#include <batchAllocator.h>
#include <string>
#include <DeepNeuralNetwork.h>

using std::cout;
using std::endl;

void run_miniMNIST_test(ClusterNet gpus)
{

	// Tests RMSprop with weight updates, logistic grad.
	// Additionally tests the interplay between different functions.

	char buff[1024] = {0};
	ssize_t len = ::readlink("/proc/self/exe", buff, sizeof(buff)-1);
	std::string path = std::string(buff);
	replace(path,"/build/testSuite.out","/tests/");

	//Matrix *X = read_hdf5((path + "/mnist_mini_X.hdf5").c_str());
	//Matrix *y = read_hdf5((path + "/mnist_mini_y.hdf5").c_str());

	Matrix *X = read_hdf5("/home/tim/data/mnist/X.hdf5");
	Matrix *y = read_hdf5("/home/tim/data/mnist/y.hdf5");


	Matrix *w1 = gpus.uniformSqrtWeight(784,1000);
	Matrix *w2 = gpus.uniformSqrtWeight(1000,10);
	Matrix *m1 = zeros(784,1000);
	Matrix *m2 = zeros(1000,10);
	Matrix *ms1 = zeros(784,1000);
	Matrix *ms2 = zeros(1000,10);
	Matrix *grad_w1_ms = zeros(784,1000);
	Matrix *grad_w2_ms = zeros(1000,10);
	Matrix *grad_w2 = empty(1000,10);
	Matrix *grad_w1 = empty(784,1000);
	float cv_error = 0.0f;
	float train_error = 0.0f;

	Matrix *z = zeros(10,1000);

	BatchAllocator b = BatchAllocator();
	b.init(X, y, 0.2, 128, 64);
	int epochs  = 20;
	float learning_rate = 0.003;
	float momentum = 0.5;
	for(int EPOCH = 1; EPOCH < epochs; EPOCH++)
	{
	  momentum += 0.01;
	  if(momentum > 0.95) momentum = 0.95;

	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {
		  b.broadcast_batch_to_processes();


		  //nesterov updates
		  scalarMul(m1,momentum,m1);
		  scalarMul(m2,momentum,m2);
		  add(w1,m1,w1);
		  add(w2,m2,w2);

		  //feedforward
		  Matrix *d0 = gpus.dropout(b.CURRENT_BATCH,0.2);
		  //print_gpus_matrix(w1);
		  Matrix *z1 = gpus.dot(d0, w1);
		  //logistic(z1, z1);
		  rectified_linear(z1,z1);

		  add_to_z(z,z1,y,10,z);

		  b.allocate_next_batch_async();

		  b.replace_current_batch_with_next();

	  }

	  scalarMul(z,1.0/(b.TOTAL_BATCHES));



	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {
		  b.broadcast_batch_to_processes();


		  //nesterov updates
		  scalarMul(m1,momentum,m1);
		  scalarMul(m2,momentum,m2);
		  add(w1,m1,w1);
		  add(w2,m2,w2);

		  //feedforward
		  Matrix *d0 = gpus.dropout(b.CURRENT_BATCH,0.2);
		  //print_gpus_matrix(w1);
		  Matrix *z1 = gpus.dot(d0, w1);
		  //logistic(z1, z1);
		  rectified_linear(z1,z1);
		  Matrix *d1 = gpus.dropout(z1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *t = create_t_matrix(b.CURRENT_BATCH_Y,10);

		  b.allocate_next_batch_async();

		  //backprop
		  Matrix *e1 = sub(out, t);
		  Matrix *e2 = gpus.dotT(e1, w2);
		  gpus.Tdot(z1,e1,grad_w2);

		  gpus.Tdot(b.CURRENT_BATCH,e2,grad_w1);

		  //weight updates
		  RMSprop_with_nesterov_weight_update(ms1,grad_w1,w1,m1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_nesterov_weight_update(ms2,grad_w2,w2,m2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);

		  hipFree(e1->data);
		  hipFree(e2->data);
		  hipFree(z1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(t->data);
		  hipFree(d0->data);
		  hipFree(d1->data);

		  b.replace_current_batch_with_next();

	  }


	  train_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {

		  b.broadcast_batch_to_processes();

		  Matrix *d0 = scalarMul(b.CURRENT_BATCH,0.8);
		  Matrix *a1 = gpus.dot(d0,w1);
		  logistic(a1, a1);
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_Y);
		  b.allocate_next_batch_async();
		  float sum_value = sum(eq);

		  train_error += (b.CURRENT_BATCH->rows - sum_value)/ (1.0f * b.CURRENT_BATCH->rows *b.TOTAL_BATCHES) ;

		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);

		  b.replace_current_batch_with_next();
	  }

	  std::cout << "Train error: " << train_error << std::endl;

	  cv_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES_CV; i++)
	  {
		  b.broadcast_batch_cv_to_processes();
		  Matrix *d0 = scalarMul(b.CURRENT_BATCH_CV,0.8);
		  Matrix *a1 = gpus.dot(d0,w1);
		  logistic(a1, a1);
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_CV_Y);
		  b.allocate_next_cv_batch_async();
		  float sum_value = sum(eq);

		  cv_error += (b.CURRENT_BATCH_CV->rows  - sum_value)/ (1.0f * b.CURRENT_BATCH_CV->rows *b.TOTAL_BATCHES_CV) ;

		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);

		  b.replace_current_cv_batch_with_next();
	  }

	  std::cout << "Cross validation error: " << cv_error << std::endl;

	}
	/*
	ASSERT(train_error < 0.03f,"mini-MNIST train error 17 epochs < 0.03.");
	ASSERT(cv_error < 0.22f, "mini-MNIST train error 17 epochs < 0.22.");

	b.finish_batch_allocator();


	Matrix *w1_dist = gpus.distributed_uniformSqrtWeight(784,1000);
	Matrix *w2_dist = gpus.distributed_uniformSqrtWeight(1000,10);
	Matrix *m1_dist = gpus.distributed_zeros(784,1000);
	Matrix *m2_dist = gpus.distributed_zeros(1000,10);
	Matrix *ms1_dist = gpus.distributed_zeros(784,1000);
	Matrix *ms2_dist = gpus.distributed_zeros(1000,10);
	Matrix *grad_w1_ms_dist = gpus.distributed_zeros(784,1000);
	Matrix *grad_w2_ms_dist = gpus.distributed_zeros(1000,10);
	Matrix *grad_w1_dist = gpus.distributed_zeros(784,1000);
	Matrix *grad_w2_dist = gpus.distributed_zeros(1000,10);

	BatchAllocator b_dist = BatchAllocator();
	b_dist.init(X, y, 0.2, 32, 64, gpus, Distributed_weights);
	for(int EPOCH = 1; EPOCH < epochs; EPOCH++)
	{
	  momentum += 0.01;
	  if(momentum > 0.95) momentum = 0.95;
	  for(int i = 0; i < b_dist.TOTAL_BATCHES; i++)
	  {

		  b_dist.broadcast_batch_to_processes();

		  //nesterov updates
		  scalarMul(m1_dist,momentum,m1_dist);
		  scalarMul(m2_dist,momentum,m2_dist);
		  add(w1_dist,m1_dist,w1_dist);
		  add(w2_dist,m2_dist,w2_dist);

		  Matrix *d0 = gpus.dropout(b_dist.CURRENT_BATCH,0.2);
		  //print_gpus_matrix(w1);
		  Matrix *z1 = gpus.dot(d0, w1_dist);
		  logistic(z1, z1);
		  Matrix *d1 = gpus.dropout(z1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2_dist);
		  Matrix *out = softmax(a2);
		  Matrix *t = create_t_matrix(b_dist.CURRENT_BATCH_Y,10);

		  b_dist.allocate_next_batch_async();

		  //backprop
		  Matrix *e1 = sub(out, t);
		  Matrix *e2 = gpus.dotT(e1, w2_dist);
		  gpus.Tdot(z1,e1,grad_w2_dist);
		  logisticGrad(z1,z1);
		  mul(e2,z1,e2);
		  gpus.Tdot(b_dist.CURRENT_BATCH,e2,grad_w1_dist);

		  RMSprop_with_nesterov_weight_update(ms1_dist,grad_w1_dist,w1_dist,m1_dist,0.9f,learning_rate,b_dist.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_nesterov_weight_update(ms2_dist,grad_w2_dist,w2_dist,m2_dist,0.9f,learning_rate,b_dist.CURRENT_BATCH->rows, momentum);

		  hipFree(e1->data);
		  hipFree(e2->data);
		  hipFree(z1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(t->data);
		  hipFree(d0->data);
		  hipFree(d1->data);

		  b_dist.replace_current_batch_with_next();

	  }

	  train_error = 0;
	  for(int i = 0; i < b_dist.TOTAL_BATCHES; i++)
	  {
		  b_dist.broadcast_batch_to_processes ();

		  Matrix *d0 = scalarMul(b_dist.CURRENT_BATCH,0.8);
		  Matrix *a1 = gpus.dot(d0,w1);

		  logistic(a1, a1);
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b_dist.CURRENT_BATCH_Y);
		  float sum_value = sum(eq);

		  b_dist.allocate_next_batch_async();

		  train_error += (b_dist.CURRENT_BATCH->rows - sum_value)/ (1.0f * b_dist.CURRENT_BATCH->rows *b_dist.TOTAL_BATCHES) ;

		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  hipFree(result->data);
		  hipFree(eq->data);

		  b_dist.replace_current_batch_with_next();
	  }

	  //std::cout << "Train error: " << train_error << std::endl;

	  cv_error = 0;
	  for(int i = 0; i < b_dist.TOTAL_BATCHES_CV; i++)
	  {
		  b_dist.broadcast_batch_cv_to_processes();

		  Matrix *d0 = scalarMul(b_dist.CURRENT_BATCH_CV,0.8);
		  Matrix *a1 = gpus.dot(d0,w1);
		  logistic(a1, a1);
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b_dist.CURRENT_BATCH_CV_Y);
		  float sum_value = sum(eq);

		  b_dist.allocate_next_cv_batch_async();

		  cv_error += (b_dist.CURRENT_BATCH_CV->rows  - sum_value)/ (1.0f * b_dist.CURRENT_BATCH_CV->rows *b_dist.TOTAL_BATCHES_CV) ;

		  hipFree(a1->data);
		  hipFree(a2->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);

		  b_dist.replace_current_cv_batch_with_next();
	  }

	  //std::cout << "Cross validation error: " << cv_error << std::endl;

	}


	ASSERT(train_error < 0.03f,"mini-MNIST train error 17 epochs < 0.03.");
	ASSERT(cv_error < 0.22f, "mini-MNIST train error 17 epochs < 0.22.");



	b_dist.finish_batch_allocator();


	// Maxout test

	// Tests RMSprop with weight updates, logistic grad.
	// Additionally tests the interplay between different functions.

	w1 = gpus.uniformSqrtWeight(784,1024);
	w2 = gpus.uniformSqrtWeight(128,10);
	m1 = zeros(784,1024);
	m2 = zeros(128,10);
	ms1 = zeros(784,1024);
	ms2 = zeros(128,10);
	grad_w1_ms = zeros(784,1024);
	grad_w2_ms = zeros(128,10);
	grad_w1 = empty(784,1024);
	grad_w2 = empty(128,10);
	cv_error = 0.0f;
	train_error = 0.0f;

	b = BatchAllocator();
	b.init(X, y, 0.2, 32, 64);
	epochs  = 17;
	learning_rate = 0.01;
	momentum = 0.5;
	for(int EPOCH = 1; EPOCH < epochs; EPOCH++)
	{
	  momentum += 0.01;
	  if(momentum > 0.95) momentum = 0.95;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {
		  b.broadcast_batch_to_processes();

		  //nesterov updates
		  scalarMul(m1,momentum,m1);
		  scalarMul(m2,momentum,m2);
		  add(w1,m1,w1);
		  add(w2,m2,w2);

		  //feedforward
		  Matrix *d0 = gpus.dropout(b.CURRENT_BATCH,0.2);
		  //print_gpus_matrix(w1);
		  Matrix *z1 = gpus.dot(d0, w1);
		  Matrix **a_paired = maxout(z1,8);
		  Matrix *a1 = a_paired[0];
		  Matrix *a1_idx = a_paired[1];
		  Matrix *d1 = gpus.dropout(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *t = create_t_matrix(b.CURRENT_BATCH_Y,10);

		  b.allocate_next_batch_async();

		  //backprop
		  Matrix *e1 = sub(out, t);
		  Matrix *e2_partial = gpus.dotT(e1, w2);
		  Matrix *e2 = empty(b.CURRENT_BATCH->rows,e2_partial->cols*8);

		  gpus.Tdot(a1,e1,grad_w2);
		  expand_to_maxout_grad(e2_partial, a1_idx,e2);
		  gpus.Tdot(b.CURRENT_BATCH,e2,grad_w1);

		  //weight updates
		  RMSprop_with_nesterov_weight_update(ms1,grad_w1,w1,m1,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);
		  RMSprop_with_nesterov_weight_update(ms2,grad_w2,w2,m2,0.9f,learning_rate,b.CURRENT_BATCH->rows, momentum);

		  hipFree(e1->data);
		  hipFree(e2->data);
		  hipFree(e2_partial->data);
		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a1_idx->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(t->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  free(a_paired);

		  b.replace_current_batch_with_next();

	  }



	  train_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES; i++)
	  {

		  b.broadcast_batch_to_processes();

		  Matrix *d0 = scalarMul(b.CURRENT_BATCH,0.8);
		  Matrix *z1 = gpus.dot(d0,w1);
		  Matrix **a1_pair = maxout(z1,8);
		  Matrix *a1 = a1_pair[0];
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_Y);
		  b.allocate_next_batch_async();
		  float sum_value = sum(eq);

		  train_error += (b.CURRENT_BATCH->rows - sum_value)/ (1.0f * b.CURRENT_BATCH->rows *b.TOTAL_BATCHES) ;

		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a1_pair[1]->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);
		  hipFree(d1->data);
		  hipFree(d0->data);
		  free(a1_pair);

		  b.replace_current_batch_with_next();
	  }

	  //std::cout << "MAXOUT Train error: " << train_error << std::endl;



	  cv_error = 0;
	  for(int i = 0; i < b.TOTAL_BATCHES_CV; i++)
	  {
		  b.broadcast_batch_cv_to_processes();
		  Matrix *d0 = scalarMul(b.CURRENT_BATCH_CV,0.8);
		  Matrix *z1 = gpus.dot(d0,w1);
		  Matrix **a1_pair = maxout(z1,8);
		  Matrix *a1 = a1_pair[0];
		  Matrix *d1 = scalarMul(a1,0.5);
		  Matrix *a2 = gpus.dot(d1,w2);
		  Matrix *out = softmax(a2);
		  Matrix *result = argmax(out);
		  Matrix *eq = equal(result,b.CURRENT_BATCH_CV_Y);
		  b.allocate_next_batch_async();
		  float sum_value = sum(eq);

		  cv_error += (b.CURRENT_BATCH_CV->rows  - sum_value)/ (1.0f * b.CURRENT_BATCH_CV->rows *b.TOTAL_BATCHES_CV) ;

		  hipFree(z1->data);
		  hipFree(a1->data);
		  hipFree(a1_pair[1]->data);
		  hipFree(a2->data);
		  hipFree(out->data);
		  hipFree(result->data);
		  hipFree(eq->data);
		  hipFree(d0->data);
		  hipFree(d1->data);
		  free(a1_pair);

		  b.replace_current_cv_batch_with_next();
	  }

	  //std::cout << "MAXOUT Cross validation error: " << cv_error << std::endl;

	}


	ASSERT(train_error < 0.02f,"mini-MNIST train error 17 epochs < 0.02.");
	ASSERT(cv_error < 0.22f, "mini-MNIST train error 17 epochs < 0.22.");
	*/
	std::vector<int> layers;
	layers.push_back(768);
	layers.push_back(512);


	BatchAllocator allocator = BatchAllocator();
	allocator.init(X,y,0.2,128,256,gpus, Distributed_weights);
	DeepNeuralNetwork net = DeepNeuralNetwork(layers,Classification, gpus, allocator, 10);
	net.EPOCHS = 1000;
	//net.LEARNING_RATE = 0.001;
	net.LEARNING_RATE = 0.001;
	net.train();

	if(gpus.MYRANK == 0)
	{
		cout << endl;
		cout << "Train error should be: 0.0025" << endl;
		cout << "Cross validation error should be: 0.13" << endl;
	}

	allocator = BatchAllocator();
	Matrix *t = to_host(create_t_matrix(to_gpu(y),10));
	allocator.init(X,t,0.2,128,256,gpus, Distributed_weights);
	net = DeepNeuralNetwork(layers,Regression, gpus, allocator, 10);
	net.EPOCHS = 100;
	net.PRINT_MISSCLASSIFICATION = true;
	net.OUTPUT_IS_PROBABILITY = true;
	net.LEARNING_RATE = 0.0003;
	net.train();

	if(gpus.MYRANK == 0)
	{
		cout << endl;
		cout << "Train error should be about: 0.05" << endl;
		cout << "Cross validation error should be about: 0.25" << endl;
	}


	/*
	if(gpus.MYGPUID == 0)
	{
		X = read_sparse_hdf5((path + "crowdflower_X_test.hdf5").c_str());
		y = read_sparse_hdf5((path + "crowdflower_y_test.hdf5").c_str());
	}
	else
	{
		X = empty_pinned_sparse(1,1,1);
		y = empty_pinned_sparse(1,1,1);
	}

	b = BatchAllocator();
	b.init(X,y,0.2,128,512,gpus, Distributed_weights_sparse);
	layers.clear();
	layers.push_back(400);
	layers.push_back(400);

	net = DeepNeuralNetwork(layers,Regression,gpus,b,24);
	net.EPOCHS = 4;
	net.TRANSITION_EPOCH = 4;
	net.LEARNING_RATE = 0.0001;
	net.OUTPUT_IS_PROBABILITY = true;
	net.train();
	*/


}
